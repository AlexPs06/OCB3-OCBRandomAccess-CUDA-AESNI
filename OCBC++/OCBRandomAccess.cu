
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int fila=0;
int columna=0;
int gris=0;
char foto[]="gato.pgm";
unsigned char imagenGlobal[5000][5000];
unsigned char imagen[5000];

class aesBlock
{
public:
    unsigned int block[4];
    unsigned int index[4];
};

unsigned char matrizCajaS[256]={
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};
void leerPGM(unsigned char imagen[],char foto[]){
    FILE *arch;
	unsigned char c,c1,c2;

	arch=fopen(foto,"rb");
	c1=fgetc(arch);
	c2=fgetc(arch);
    if (c1!='P' || c2!='5'){
		printf("\nFormato no corresponde a una Imagen\n");
		exit (0);
    }
	c=fgetc(arch);
	fscanf(arch,"%d",&columna);
    c=fgetc(arch);

	fscanf(arch,"%d",&fila);
    while (c!='\n')
		c=fgetc(arch);
	fscanf(arch,"%d",&gris);

    cout<<c1<<c2<<endl;
    cout<<"fila "<<fila<<endl;
    cout<<"columna "<<columna<<endl;
    cout<<"gris "<<gris<<endl;
    int k = 0;
	for(int i=0; i<fila; i++)
		for(int j=0; j<columna; j++)
		{
			c=fgetc(arch);
            int temp = (int)c ;
            imagenGlobal[i][j] = temp;
            imagen[k]=(unsigned char)c;
            k++;
		}
        fclose(arch);
        
}

void ExpansionKeys128(const unsigned int *k,unsigned long long klen,  unsigned int keys[11][4] ){
    unsigned char RotWordTemp[4];
    const unsigned int matrizRcon[10]={ 0x01000000, 0x02000000, 0x04000000, 0x08000000, 0x10000000, 0x20000000, 0x40000000, 0x80000000, 0x1b000000, 0x36000000};
    // *keys[0] =  *k;
    memcpy(&keys[0], k, 16);
    for(int i = 0; i<10; i++){
        RotWordTemp[0]=keys[i][3]>>16;
        RotWordTemp[1]=keys[i][3]>>8;
        RotWordTemp[2]=keys[i][3];
        RotWordTemp[3]=keys[i][3]>>24; 

       

        
        for(int j = 0;  j < 4; j++ ){
            RotWordTemp[j] = matrizCajaS[ (int) RotWordTemp[j] ];
        }
        int RotWord = 0;
        RotWord = RotWord ^ ( (int) RotWordTemp[0])<<24;
        RotWord = RotWord ^ ( (int) RotWordTemp[1])<<16;
        RotWord = RotWord ^ ( (int) RotWordTemp[2])<<8;
        RotWord = RotWord ^ ( (int) RotWordTemp[3]);
        
        keys[i+1][0] =  RotWord ^ keys[i][0];
        keys[i+1][0] = keys[i+1][0] ^ matrizRcon[i];
        
        for(int x = 1;  x < 4; x++ ){
            for(int j = 0;  j < 4; j++){
                keys[i+1][(j)+x ] =  keys[i+1][j+x-1] ^ keys[i][(j)+x];
            }
        }
        
    }

}
void imprimiArreglo(int tam, unsigned int *keys ){
    
    for (int i = 0; i<tam; i++){
        // cout<< hex(keys[i]) <<" ";
        
        printf("%x \n", keys[i] );
        
    }
    printf("\n---------------------------\n");
}


__device__ void imprimiArregloCuda(int tam,unsigned char *keys ){
    for (int i = 0; i<tam; i++){
        // cout<< hex(keys[i]) <<" ";
        if(i%4==0)
            printf("\n");
        printf("%x ", keys[i] & 0xff);
        
    }
}
__device__ void imprimiArregloCudaInt(int tam,unsigned int *keys ){
    printf("----------------\n" );
    
    for (int i = 0; i<tam; i++){
        // cout<< hex(keys[i]) <<" ";
      
        printf("%x \n", keys[i] );
        
    }
}
__device__ void XOR_128(unsigned int* A, unsigned int * B )
{
	for (int i = 0; i < 4; i++) { 
        A[i] =A[i] ^  B[i];
    }
}

__device__ void addRoundKey(unsigned int* in, unsigned int * keys, int round)
{

	for (int i = 0; i < 4; i++) { 
        in[i] =in[i] ^  keys[(round*4) + i];
    }

}

__device__ void subBytes(unsigned int* in, unsigned char * matrizCajaS)
{
    unsigned char * temp;
    temp = (unsigned char *) in;
	for (int i = 0; i < 16; i++) { 
        temp[i] = matrizCajaS[ (int) temp[i]]; 
    }
}



__device__ void shiftRows(unsigned int* in, int *shifttab){
    
    unsigned char * temp;
    

    unsigned char h[16];
    temp = (unsigned char *) in;
	
    memcpy(h, temp, 16);

    for(int i = 0; i < 16; i++){
        temp[i] = h[shifttab[i]];
    }
}
__device__ unsigned char GF2Redution(unsigned short in ){
    
    unsigned short temp = in ; // numero de prueba
    // temp = temp * 2;
    unsigned char temp1 = temp>>8; //queda FF
    unsigned char temp2 = temp1<<1; //queda FE
    unsigned char temp3 = temp1<<3; //queda F8
    unsigned char temp4 = temp1<<4; //queda F

    unsigned char temp5 = temp>>13;//queda 7
    unsigned char temp6 = temp5<<1;//queda E 
    unsigned char temp7 = temp5<<3;//queda 38
    unsigned char temp8 = temp5<<4;//queda 70

    unsigned char temp9 = temp>>12;//queda F
    unsigned char temp10 = temp9<<1;//queda 1E 
    unsigned char temp11 = temp9<<3;//queda 78
    unsigned char temp12 = temp9<<4;//queda F0
    
    in = temp ^ temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6 ^ temp7 ^ temp8 ^ temp9 ^ temp10 ^ temp11 ^ temp12; 
    return in;
}

__device__ unsigned char multiplicacionENGF2(int caso , unsigned short numero2){
    switch (caso)
    {
    case 1:
        return numero2;
        break;
    case 2:
        numero2 = GF2Redution(numero2 * 2); 
        return numero2 ;
        break;
    case 3:
        numero2 = GF2Redution((numero2*2)^numero2); 
        return numero2;
        break;
    default:
        break;
    }
    return 0;

}

__device__ void mixColumns(unsigned char* in ){
    // unsigned char T1[4]={in[0],in[4],in[8],in[12] };
    // unsigned char T2[4]={in[1],in[5],in[9],in[13] };
    // unsigned char T3[4]={in[2],in[6],in[10],in[14] };
    // unsigned char T4[4]={in[3],in[7],in[11],in[15] };

    for (int i = 0; i < 4; i++){
        unsigned char T1[4]={in[0+i],in[0+i],in[0+i],in[0+i] };
        unsigned char T2[4]={in[4+i],in[4+i],in[4+i],in[4+i] };
        unsigned char T3[4]={in[8+i],in[8+i],in[8+i],in[8+i] };
        unsigned char T4[4]={in[12+i],in[12+i],in[12+i],in[12+i] };
    
        T1[0] =  multiplicacionENGF2(2, T1[0]);
        T1[3] =  multiplicacionENGF2(3, T1[3]);

        T2[0] =  multiplicacionENGF2(3, T2[0]);
        T2[1] =  multiplicacionENGF2(2, T2[1]);

        T3[1] =  multiplicacionENGF2(3, T3[1]);
        T3[2] =  multiplicacionENGF2(2, T3[2]);

        T4[2] =  multiplicacionENGF2(3, T4[2]);
        T4[3] =  multiplicacionENGF2(2, T4[3]);

        
        T1[0] = T1[0] ^ T2[0] ^ T3[0] ^ T4[0];
        T1[1] = T1[1] ^ T2[1] ^ T3[1] ^ T4[1];
        T1[2] = T1[2] ^ T2[2] ^ T3[2] ^ T4[2];
        T1[3] = T1[3] ^ T2[3] ^ T3[3] ^ T4[3];
        


        in[0+i] = T1[0];  
        in[4+i] = T1[1];
        in[8+i] = T1[2]; 
        in[12+i] = T1[3];
    }
    
}

__device__ void subBytesMixColumns(unsigned int* in, int * T1, int * T2, int * T3, int * T4){
	unsigned char * temp;
    temp = (unsigned char *) in;
    for (int i = 0; i < 16; i=i+4){
        // unsigned char tempT1[4]={ T1[(int) in[0+i]] >> 24 & ff   , T1[(int) in[0+i]] >> 16 ,T1[(int) in[0+i]] >> 8 , T1[(int) in[0+i]] };
        // unsigned char tempT11[4] = { (unsigned char) (T1[(int) in[0+i]] >> 24), (unsigned char) (T1[(int) in[0+i]] >> 16), (unsigned char) (T1[(int) in[0+i]] >> 8), (unsigned char) (T1[(int) in[0+i] ])  }; 
        // unsigned char tempT12[4]={ (unsigned char) (T2[(int) in[4+i]] >> 24), (unsigned char) (T2[(int) in[4+i]] >> 16), (unsigned char) (T2[(int) in[4+i]] >> 8), (unsigned char) (T2[(int) in[4+i] ]) };
        // unsigned char tempT13[4]={ (unsigned char) (T3[(int) in[8+i]] >> 24), (unsigned char) (T3[(int) in[8+i]] >> 16), (unsigned char) (T3[(int) in[8+i]] >> 8), (unsigned char) (T3[(int) in[8+i] ]) };
        // unsigned char tempT14[4]={ (unsigned char) (T4[(int) in[12+i]] >> 24), (unsigned char) (T4[(int) in[12+i]] >> 16), (unsigned char) (T4[(int) in[12+i]] >> 8), (unsigned char) (T4[(int) in[12+i] ]) };
        
        int tempT1 = T1[(int) temp[i+3]];
        int tempT2 = T2[(int) temp[i+2]];
        int tempT3 = T3[(int) temp[i+1]];
        int tempT4 = T4[(int) temp[i] ];

        // printf("%x \n", temp[i+3] );
        // printf("%x \n", temp[i+2] );
        // printf("%x \n", temp[i+1] );
        // printf("%x \n", temp[i] );

        int tempT5 = tempT1 ^ tempT2 ^ tempT3 ^ tempT4; 
        // printf("%x \n", tempT5 );
        in[i/4] = tempT5;
        // printf("%x \n", in[i] );
        // tempT11[0] = tempT11[0] ^ tempT12[0] ^ tempT13[0] ^ tempT14[0];
        // tempT11[1] = tempT11[1] ^ tempT12[1] ^ tempT13[1] ^ tempT14[1];
        // tempT11[2] = tempT11[2] ^ tempT12[2] ^ tempT13[2] ^ tempT14[2];
        // tempT11[3] = tempT11[3] ^ tempT12[3] ^ tempT13[3] ^ tempT14[3];
    }
    // imprimiArregloCudaInt(4,in );
}

__device__ void AES_init( unsigned char  *matrizCajaS, int *T1, int *T2, int *T3, int *T4){
    unsigned char matrizCajaSTemp[256]={
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
    };
    int T1Temp[256] ={
        0xc66363a5, 0xf87c7c84, 0xee777799, 0xf67b7b8d, 0xfff2f20d, 0xd66b6bbd, 0xde6f6fb1, 0x91c5c554, 0x60303050, 0x2010103, 0xce6767a9, 0x562b2b7d, 0xe7fefe19, 0xb5d7d762, 0x4dababe6, 0xec76769a, 
        0x8fcaca45, 0x1f82829d, 0x89c9c940, 0xfa7d7d87, 0xeffafa15, 0xb25959eb, 0x8e4747c9, 0xfbf0f00b, 0x41adadec, 0xb3d4d467, 0x5fa2a2fd, 0x45afafea, 0x239c9cbf, 0x53a4a4f7, 0xe4727296, 0x9bc0c05b, 
        0x75b7b7c2, 0xe1fdfd1c, 0x3d9393ae, 0x4c26266a, 0x6c36365a, 0x7e3f3f41, 0xf5f7f702, 0x83cccc4f, 0x6834345c, 0x51a5a5f4, 0xd1e5e534, 0xf9f1f108, 0xe2717193, 0xabd8d873, 0x62313153, 0x2a15153f, 
        0x804040c, 0x95c7c752, 0x46232365, 0x9dc3c35e, 0x30181828, 0x379696a1, 0xa05050f, 0x2f9a9ab5, 0xe070709, 0x24121236, 0x1b80809b, 0xdfe2e23d, 0xcdebeb26, 0x4e272769, 0x7fb2b2cd, 0xea75759f, 
        0x1209091b, 0x1d83839e, 0x582c2c74, 0x341a1a2e, 0x361b1b2d, 0xdc6e6eb2, 0xb45a5aee, 0x5ba0a0fb, 0xa45252f6, 0x763b3b4d, 0xb7d6d661, 0x7db3b3ce, 0x5229297b, 0xdde3e33e, 0x5e2f2f71, 0x13848497, 
        0xa65353f5, 0xb9d1d168, 0x0, 0xc1eded2c, 0x40202060, 0xe3fcfc1f, 0x79b1b1c8, 0xb65b5bed, 0xd46a6abe, 0x8dcbcb46, 0x67bebed9, 0x7239394b, 0x944a4ade, 0x984c4cd4, 0xb05858e8, 0x85cfcf4a, 
        0xbbd0d06b, 0xc5efef2a, 0x4faaaae5, 0xedfbfb16, 0x864343c5, 0x9a4d4dd7, 0x66333355, 0x11858594, 0x8a4545cf, 0xe9f9f910, 0x4020206, 0xfe7f7f81, 0xa05050f0, 0x783c3c44, 0x259f9fba, 0x4ba8a8e3, 
        0xa25151f3, 0x5da3a3fe, 0x804040c0, 0x58f8f8a, 0x3f9292ad, 0x219d9dbc, 0x70383848, 0xf1f5f504, 0x63bcbcdf, 0x77b6b6c1, 0xafdada75, 0x42212163, 0x20101030, 0xe5ffff1a, 0xfdf3f30e, 0xbfd2d26d, 
        0x81cdcd4c, 0x180c0c14, 0x26131335, 0xc3ecec2f, 0xbe5f5fe1, 0x359797a2, 0x884444cc, 0x2e171739, 0x93c4c457, 0x55a7a7f2, 0xfc7e7e82, 0x7a3d3d47, 0xc86464ac, 0xba5d5de7, 0x3219192b, 0xe6737395, 
        0xc06060a0, 0x19818198, 0x9e4f4fd1, 0xa3dcdc7f, 0x44222266, 0x542a2a7e, 0x3b9090ab, 0xb888883, 0x8c4646ca, 0xc7eeee29, 0x6bb8b8d3, 0x2814143c, 0xa7dede79, 0xbc5e5ee2, 0x160b0b1d, 0xaddbdb76, 
        0xdbe0e03b, 0x64323256, 0x743a3a4e, 0x140a0a1e, 0x924949db, 0xc06060a, 0x4824246c, 0xb85c5ce4, 0x9fc2c25d, 0xbdd3d36e, 0x43acacef, 0xc46262a6, 0x399191a8, 0x319595a4, 0xd3e4e437, 0xf279798b, 
        0xd5e7e732, 0x8bc8c843, 0x6e373759, 0xda6d6db7, 0x18d8d8c, 0xb1d5d564, 0x9c4e4ed2, 0x49a9a9e0, 0xd86c6cb4, 0xac5656fa, 0xf3f4f407, 0xcfeaea25, 0xca6565af, 0xf47a7a8e, 0x47aeaee9, 0x10080818, 
        0x6fbabad5, 0xf0787888, 0x4a25256f, 0x5c2e2e72, 0x381c1c24, 0x57a6a6f1, 0x73b4b4c7, 0x97c6c651, 0xcbe8e823, 0xa1dddd7c, 0xe874749c, 0x3e1f1f21, 0x964b4bdd, 0x61bdbddc, 0xd8b8b86, 0xf8a8a85, 
        0xe0707090, 0x7c3e3e42, 0x71b5b5c4, 0xcc6666aa, 0x904848d8, 0x6030305, 0xf7f6f601, 0x1c0e0e12, 0xc26161a3, 0x6a35355f, 0xae5757f9, 0x69b9b9d0, 0x17868691, 0x99c1c158, 0x3a1d1d27, 0x279e9eb9, 
        0xd9e1e138, 0xebf8f813, 0x2b9898b3, 0x22111133, 0xd26969bb, 0xa9d9d970, 0x78e8e89, 0x339494a7, 0x2d9b9bb6, 0x3c1e1e22, 0x15878792, 0xc9e9e920, 0x87cece49, 0xaa5555ff, 0x50282878, 0xa5dfdf7a, 
        0x38c8c8f, 0x59a1a1f8, 0x9898980, 0x1a0d0d17, 0x65bfbfda, 0xd7e6e631, 0x844242c6, 0xd06868b8, 0x824141c3, 0x299999b0, 0x5a2d2d77, 0x1e0f0f11, 0x7bb0b0cb, 0xa85454fc, 0x6dbbbbd6, 0x2c16163a
    };
   
    int T2Temp[256] = {
        0xa5c66363, 0x84f87c7c, 0x99ee7777, 0x8df67b7b, 0xdfff2f2, 0xbdd66b6b, 0xb1de6f6f, 0x5491c5c5, 0x50603030, 0x3020101, 0xa9ce6767, 0x7d562b2b, 0x19e7fefe, 0x62b5d7d7, 0xe64dabab, 0x9aec7676, 
        0x458fcaca, 0x9d1f8282, 0x4089c9c9, 0x87fa7d7d, 0x15effafa, 0xebb25959, 0xc98e4747, 0xbfbf0f0, 0xec41adad, 0x67b3d4d4, 0xfd5fa2a2, 0xea45afaf, 0xbf239c9c, 0xf753a4a4, 0x96e47272, 0x5b9bc0c0, 
        0xc275b7b7, 0x1ce1fdfd, 0xae3d9393, 0x6a4c2626, 0x5a6c3636, 0x417e3f3f, 0x2f5f7f7, 0x4f83cccc, 0x5c683434, 0xf451a5a5, 0x34d1e5e5, 0x8f9f1f1, 0x93e27171, 0x73abd8d8, 0x53623131, 0x3f2a1515, 
        0xc080404, 0x5295c7c7, 0x65462323, 0x5e9dc3c3, 0x28301818, 0xa1379696, 0xf0a0505, 0xb52f9a9a, 0x90e0707, 0x36241212, 0x9b1b8080, 0x3ddfe2e2, 0x26cdebeb, 0x694e2727, 0xcd7fb2b2, 0x9fea7575, 
        0x1b120909, 0x9e1d8383, 0x74582c2c, 0x2e341a1a, 0x2d361b1b, 0xb2dc6e6e, 0xeeb45a5a, 0xfb5ba0a0, 0xf6a45252, 0x4d763b3b, 0x61b7d6d6, 0xce7db3b3, 0x7b522929, 0x3edde3e3, 0x715e2f2f, 0x97138484, 
        0xf5a65353, 0x68b9d1d1, 0x0, 0x2cc1eded, 0x60402020, 0x1fe3fcfc, 0xc879b1b1, 0xedb65b5b, 0xbed46a6a, 0x468dcbcb, 0xd967bebe, 0x4b723939, 0xde944a4a, 0xd4984c4c, 0xe8b05858, 0x4a85cfcf, 
        0x6bbbd0d0, 0x2ac5efef, 0xe54faaaa, 0x16edfbfb, 0xc5864343, 0xd79a4d4d, 0x55663333, 0x94118585, 0xcf8a4545, 0x10e9f9f9, 0x6040202, 0x81fe7f7f, 0xf0a05050, 0x44783c3c, 0xba259f9f, 0xe34ba8a8, 
        0xf3a25151, 0xfe5da3a3, 0xc0804040, 0x8a058f8f, 0xad3f9292, 0xbc219d9d, 0x48703838, 0x4f1f5f5, 0xdf63bcbc, 0xc177b6b6, 0x75afdada, 0x63422121, 0x30201010, 0x1ae5ffff, 0xefdf3f3, 0x6dbfd2d2, 
        0x4c81cdcd, 0x14180c0c, 0x35261313, 0x2fc3ecec, 0xe1be5f5f, 0xa2359797, 0xcc884444, 0x392e1717, 0x5793c4c4, 0xf255a7a7, 0x82fc7e7e, 0x477a3d3d, 0xacc86464, 0xe7ba5d5d, 0x2b321919, 0x95e67373, 
        0xa0c06060, 0x98198181, 0xd19e4f4f, 0x7fa3dcdc, 0x66442222, 0x7e542a2a, 0xab3b9090, 0x830b8888, 0xca8c4646, 0x29c7eeee, 0xd36bb8b8, 0x3c281414, 0x79a7dede, 0xe2bc5e5e, 0x1d160b0b, 0x76addbdb, 
        0x3bdbe0e0, 0x56643232, 0x4e743a3a, 0x1e140a0a, 0xdb924949, 0xa0c0606, 0x6c482424, 0xe4b85c5c, 0x5d9fc2c2, 0x6ebdd3d3, 0xef43acac, 0xa6c46262, 0xa8399191, 0xa4319595, 0x37d3e4e4, 0x8bf27979, 
        0x32d5e7e7, 0x438bc8c8, 0x596e3737, 0xb7da6d6d, 0x8c018d8d, 0x64b1d5d5, 0xd29c4e4e, 0xe049a9a9, 0xb4d86c6c, 0xfaac5656, 0x7f3f4f4, 0x25cfeaea, 0xafca6565, 0x8ef47a7a, 0xe947aeae, 0x18100808, 
        0xd56fbaba, 0x88f07878, 0x6f4a2525, 0x725c2e2e, 0x24381c1c, 0xf157a6a6, 0xc773b4b4, 0x5197c6c6, 0x23cbe8e8, 0x7ca1dddd, 0x9ce87474, 0x213e1f1f, 0xdd964b4b, 0xdc61bdbd, 0x860d8b8b, 0x850f8a8a, 
        0x90e07070, 0x427c3e3e, 0xc471b5b5, 0xaacc6666, 0xd8904848, 0x5060303, 0x1f7f6f6, 0x121c0e0e, 0xa3c26161, 0x5f6a3535, 0xf9ae5757, 0xd069b9b9, 0x91178686, 0x5899c1c1, 0x273a1d1d, 0xb9279e9e, 
        0x38d9e1e1, 0x13ebf8f8, 0xb32b9898, 0x33221111, 0xbbd26969, 0x70a9d9d9, 0x89078e8e, 0xa7339494, 0xb62d9b9b, 0x223c1e1e, 0x92158787, 0x20c9e9e9, 0x4987cece, 0xffaa5555, 0x78502828, 0x7aa5dfdf, 
        0x8f038c8c, 0xf859a1a1, 0x80098989, 0x171a0d0d, 0xda65bfbf, 0x31d7e6e6, 0xc6844242, 0xb8d06868, 0xc3824141, 0xb0299999, 0x775a2d2d, 0x111e0f0f, 0xcb7bb0b0, 0xfca85454, 0xd66dbbbb, 0x3a2c1616
    };
    int T3Temp[256] = {
        0x63a5c663, 0x7c84f87c, 0x7799ee77, 0x7b8df67b, 0xf20dfff2, 0x6bbdd66b, 0x6fb1de6f, 0xc55491c5, 0x30506030, 0x1030201, 0x67a9ce67, 0x2b7d562b, 0xfe19e7fe, 0xd762b5d7, 0xabe64dab, 0x769aec76, 
        0xca458fca, 0x829d1f82, 0xc94089c9, 0x7d87fa7d, 0xfa15effa, 0x59ebb259, 0x47c98e47, 0xf00bfbf0, 0xadec41ad, 0xd467b3d4, 0xa2fd5fa2, 0xafea45af, 0x9cbf239c, 0xa4f753a4, 0x7296e472, 0xc05b9bc0, 
        0xb7c275b7, 0xfd1ce1fd, 0x93ae3d93, 0x266a4c26, 0x365a6c36, 0x3f417e3f, 0xf702f5f7, 0xcc4f83cc, 0x345c6834, 0xa5f451a5, 0xe534d1e5, 0xf108f9f1, 0x7193e271, 0xd873abd8, 0x31536231, 0x153f2a15, 
        0x40c0804, 0xc75295c7, 0x23654623, 0xc35e9dc3, 0x18283018, 0x96a13796, 0x50f0a05, 0x9ab52f9a, 0x7090e07, 0x12362412, 0x809b1b80, 0xe23ddfe2, 0xeb26cdeb, 0x27694e27, 0xb2cd7fb2, 0x759fea75, 
        0x91b1209, 0x839e1d83, 0x2c74582c, 0x1a2e341a, 0x1b2d361b, 0x6eb2dc6e, 0x5aeeb45a, 0xa0fb5ba0, 0x52f6a452, 0x3b4d763b, 0xd661b7d6, 0xb3ce7db3, 0x297b5229, 0xe33edde3, 0x2f715e2f, 0x84971384, 
        0x53f5a653, 0xd168b9d1, 0x0, 0xed2cc1ed, 0x20604020, 0xfc1fe3fc, 0xb1c879b1, 0x5bedb65b, 0x6abed46a, 0xcb468dcb, 0xbed967be, 0x394b7239, 0x4ade944a, 0x4cd4984c, 0x58e8b058, 0xcf4a85cf, 
        0xd06bbbd0, 0xef2ac5ef, 0xaae54faa, 0xfb16edfb, 0x43c58643, 0x4dd79a4d, 0x33556633, 0x85941185, 0x45cf8a45, 0xf910e9f9, 0x2060402, 0x7f81fe7f, 0x50f0a050, 0x3c44783c, 0x9fba259f, 0xa8e34ba8, 
        0x51f3a251, 0xa3fe5da3, 0x40c08040, 0x8f8a058f, 0x92ad3f92, 0x9dbc219d, 0x38487038, 0xf504f1f5, 0xbcdf63bc, 0xb6c177b6, 0xda75afda, 0x21634221, 0x10302010, 0xff1ae5ff, 0xf30efdf3, 0xd26dbfd2, 
        0xcd4c81cd, 0xc14180c, 0x13352613, 0xec2fc3ec, 0x5fe1be5f, 0x97a23597, 0x44cc8844, 0x17392e17, 0xc45793c4, 0xa7f255a7, 0x7e82fc7e, 0x3d477a3d, 0x64acc864, 0x5de7ba5d, 0x192b3219, 0x7395e673, 
        0x60a0c060, 0x81981981, 0x4fd19e4f, 0xdc7fa3dc, 0x22664422, 0x2a7e542a, 0x90ab3b90, 0x88830b88, 0x46ca8c46, 0xee29c7ee, 0xb8d36bb8, 0x143c2814, 0xde79a7de, 0x5ee2bc5e, 0xb1d160b, 0xdb76addb, 
        0xe03bdbe0, 0x32566432, 0x3a4e743a, 0xa1e140a, 0x49db9249, 0x60a0c06, 0x246c4824, 0x5ce4b85c, 0xc25d9fc2, 0xd36ebdd3, 0xacef43ac, 0x62a6c462, 0x91a83991, 0x95a43195, 0xe437d3e4, 0x798bf279, 
        0xe732d5e7, 0xc8438bc8, 0x37596e37, 0x6db7da6d, 0x8d8c018d, 0xd564b1d5, 0x4ed29c4e, 0xa9e049a9, 0x6cb4d86c, 0x56faac56, 0xf407f3f4, 0xea25cfea, 0x65afca65, 0x7a8ef47a, 0xaee947ae, 0x8181008, 
        0xbad56fba, 0x7888f078, 0x256f4a25, 0x2e725c2e, 0x1c24381c, 0xa6f157a6, 0xb4c773b4, 0xc65197c6, 0xe823cbe8, 0xdd7ca1dd, 0x749ce874, 0x1f213e1f, 0x4bdd964b, 0xbddc61bd, 0x8b860d8b, 0x8a850f8a, 
        0x7090e070, 0x3e427c3e, 0xb5c471b5, 0x66aacc66, 0x48d89048, 0x3050603, 0xf601f7f6, 0xe121c0e, 0x61a3c261, 0x355f6a35, 0x57f9ae57, 0xb9d069b9, 0x86911786, 0xc15899c1, 0x1d273a1d, 0x9eb9279e, 
        0xe138d9e1, 0xf813ebf8, 0x98b32b98, 0x11332211, 0x69bbd269, 0xd970a9d9, 0x8e89078e, 0x94a73394, 0x9bb62d9b, 0x1e223c1e, 0x87921587, 0xe920c9e9, 0xce4987ce, 0x55ffaa55, 0x28785028, 0xdf7aa5df, 
        0x8c8f038c, 0xa1f859a1, 0x89800989, 0xd171a0d, 0xbfda65bf, 0xe631d7e6, 0x42c68442, 0x68b8d068, 0x41c38241, 0x99b02999, 0x2d775a2d, 0xf111e0f, 0xb0cb7bb0, 0x54fca854, 0xbbd66dbb, 0x163a2c16 
    };
    int T4Temp[256] = {
        0x6363a5c6, 0x7c7c84f8, 0x777799ee, 0x7b7b8df6, 0xf2f20dff, 0x6b6bbdd6, 0x6f6fb1de, 0xc5c55491, 0x30305060, 0x1010302, 0x6767a9ce, 0x2b2b7d56, 0xfefe19e7, 0xd7d762b5, 0xababe64d, 0x76769aec, 
        0xcaca458f, 0x82829d1f, 0xc9c94089, 0x7d7d87fa, 0xfafa15ef, 0x5959ebb2, 0x4747c98e, 0xf0f00bfb, 0xadadec41, 0xd4d467b3, 0xa2a2fd5f, 0xafafea45, 0x9c9cbf23, 0xa4a4f753, 0x727296e4, 0xc0c05b9b, 
        0xb7b7c275, 0xfdfd1ce1, 0x9393ae3d, 0x26266a4c, 0x36365a6c, 0x3f3f417e, 0xf7f702f5, 0xcccc4f83, 0x34345c68, 0xa5a5f451, 0xe5e534d1, 0xf1f108f9, 0x717193e2, 0xd8d873ab, 0x31315362, 0x15153f2a, 
        0x4040c08, 0xc7c75295, 0x23236546, 0xc3c35e9d, 0x18182830, 0x9696a137, 0x5050f0a, 0x9a9ab52f, 0x707090e, 0x12123624, 0x80809b1b, 0xe2e23ddf, 0xebeb26cd, 0x2727694e, 0xb2b2cd7f, 0x75759fea, 
        0x9091b12, 0x83839e1d, 0x2c2c7458, 0x1a1a2e34, 0x1b1b2d36, 0x6e6eb2dc, 0x5a5aeeb4, 0xa0a0fb5b, 0x5252f6a4, 0x3b3b4d76, 0xd6d661b7, 0xb3b3ce7d, 0x29297b52, 0xe3e33edd, 0x2f2f715e, 0x84849713, 
        0x5353f5a6, 0xd1d168b9, 0x0, 0xeded2cc1, 0x20206040, 0xfcfc1fe3, 0xb1b1c879, 0x5b5bedb6, 0x6a6abed4, 0xcbcb468d, 0xbebed967, 0x39394b72, 0x4a4ade94, 0x4c4cd498, 0x5858e8b0, 0xcfcf4a85, 
        0xd0d06bbb, 0xefef2ac5, 0xaaaae54f, 0xfbfb16ed, 0x4343c586, 0x4d4dd79a, 0x33335566, 0x85859411, 0x4545cf8a, 0xf9f910e9, 0x2020604, 0x7f7f81fe, 0x5050f0a0, 0x3c3c4478, 0x9f9fba25, 0xa8a8e34b, 
        0x5151f3a2, 0xa3a3fe5d, 0x4040c080, 0x8f8f8a05, 0x9292ad3f, 0x9d9dbc21, 0x38384870, 0xf5f504f1, 0xbcbcdf63, 0xb6b6c177, 0xdada75af, 0x21216342, 0x10103020, 0xffff1ae5, 0xf3f30efd, 0xd2d26dbf, 
        0xcdcd4c81, 0xc0c1418, 0x13133526, 0xecec2fc3, 0x5f5fe1be, 0x9797a235, 0x4444cc88, 0x1717392e, 0xc4c45793, 0xa7a7f255, 0x7e7e82fc, 0x3d3d477a, 0x6464acc8, 0x5d5de7ba, 0x19192b32, 0x737395e6, 
        0x6060a0c0, 0x81819819, 0x4f4fd19e, 0xdcdc7fa3, 0x22226644, 0x2a2a7e54, 0x9090ab3b, 0x8888830b, 0x4646ca8c, 0xeeee29c7, 0xb8b8d36b, 0x14143c28, 0xdede79a7, 0x5e5ee2bc, 0xb0b1d16, 0xdbdb76ad, 
        0xe0e03bdb, 0x32325664, 0x3a3a4e74, 0xa0a1e14, 0x4949db92, 0x6060a0c, 0x24246c48, 0x5c5ce4b8, 0xc2c25d9f, 0xd3d36ebd, 0xacacef43, 0x6262a6c4, 0x9191a839, 0x9595a431, 0xe4e437d3, 0x79798bf2, 
        0xe7e732d5, 0xc8c8438b, 0x3737596e, 0x6d6db7da, 0x8d8d8c01, 0xd5d564b1, 0x4e4ed29c, 0xa9a9e049, 0x6c6cb4d8, 0x5656faac, 0xf4f407f3, 0xeaea25cf, 0x6565afca, 0x7a7a8ef4, 0xaeaee947, 0x8081810, 
        0xbabad56f, 0x787888f0, 0x25256f4a, 0x2e2e725c, 0x1c1c2438, 0xa6a6f157, 0xb4b4c773, 0xc6c65197, 0xe8e823cb, 0xdddd7ca1, 0x74749ce8, 0x1f1f213e, 0x4b4bdd96, 0xbdbddc61, 0x8b8b860d, 0x8a8a850f, 
        0x707090e0, 0x3e3e427c, 0xb5b5c471, 0x6666aacc, 0x4848d890, 0x3030506, 0xf6f601f7, 0xe0e121c, 0x6161a3c2, 0x35355f6a, 0x5757f9ae, 0xb9b9d069, 0x86869117, 0xc1c15899, 0x1d1d273a, 0x9e9eb927, 
        0xe1e138d9, 0xf8f813eb, 0x9898b32b, 0x11113322, 0x6969bbd2, 0xd9d970a9, 0x8e8e8907, 0x9494a733, 0x9b9bb62d, 0x1e1e223c, 0x87879215, 0xe9e920c9, 0xcece4987, 0x5555ffaa, 0x28287850, 0xdfdf7aa5, 
        0x8c8c8f03, 0xa1a1f859, 0x89898009, 0xd0d171a, 0xbfbfda65, 0xe6e631d7, 0x4242c684, 0x6868b8d0, 0x4141c382, 0x9999b029, 0x2d2d775a, 0xf0f111e, 0xb0b0cb7b, 0x5454fca8, 0xbbbbd66d, 0x16163a2c
    };
    // T1 = T1Temp;
    // T2 = T2Temp;
    // T3 = T3Temp;
    // T4 = T4Temp;
    for(int i=0; i<256;i++){
        matrizCajaS[i] = matrizCajaSTemp[i];
        T1[i] = T1Temp[i];
        T2[i] = T2Temp[i];
        T3[i] = T3Temp[i];
        T4[i] = T4Temp[i];
    }
    // printf("%x estoy en init\n",&matrizCajaS[0]);
    // printf("%x\n",&matrizCajaSTemp);
}

__device__ void AES_128(aesBlock *m, unsigned long long mlen, unsigned int *keys, int index){
    // printf("%i \n",threadIdx.x);
    __shared__ unsigned char matrizCajaS[256];
    __shared__ int T1[256]; 
    __shared__ int T2[256]; 
    __shared__ int T3[256]; 
    __shared__ int T4[256];
        if(threadIdx.x == 0 ){
            AES_init(matrizCajaS, T1, T2, T3, T4);
            // printf("%x estoy en encrypt\n",&matrizCajaS[0]);
            // imprimiArregloCuda(256,matrizCajaS );
        }
        
        int shifttab[16]= {
            12, 9, 6, 3,   
            0, 13, 10, 7,  
            4, 1, 14, 11,
            8, 5, 2, 15 
            };
        
        __syncthreads();
        unsigned int block[4];

        
        // memcpy(block,)
        for (int i = 0 ; i< 4 ; i++){
            block[i]= m[index].block[i];
        }
        
        
        addRoundKey( block, keys,0);
        // // imprimiArregloCuda(16,block );
        // // imprimiArregloCuda(16,&keys[0] );

        for (int j = 1; j < 10; j++){
            // subBytes(block, matrizCajaS);
            shiftRows(block, shifttab);
            //mixColumns(block);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);
            // imprimiArregloCudaInt(4,block );
            addRoundKey( block, keys,j); // 
        }
        subBytes(block, matrizCajaS);

        shiftRows(block, shifttab);


        addRoundKey( block, keys,10);
        for (int i = 0 ; i< 4 ; i++){
            m[index].block[i]= block[i];
        }
        // imprimiArregloCudaInt(4,result );
}

__device__ void OCBAESDelta2Rounds(unsigned int block[4],  unsigned int *keys){
    // printf("%i \n",threadIdx.x);
        unsigned char matrizCajaS[256];
        int T1[256]; 
        int T2[256]; 
        int T3[256]; 
        int T4[256];
            AES_init(matrizCajaS, T1, T2, T3, T4);
        int shifttab[16]= {
            12, 9, 6, 3,   
            0, 13, 10, 7,  
            4, 1, 14, 11,
            8, 5, 2, 15 
            };
        addRoundKey( block, keys,0);
        for (int j = 1; j < 3; j++){
            shiftRows(block, shifttab);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);
            addRoundKey( block, keys,j); // 
        }
        subBytes(block, matrizCajaS);
        shiftRows(block, shifttab);
        addRoundKey( block, keys,10);
}

__global__ void Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    // int index2 = blockDim.x*blockIdx.x;
    
    
    if( index<mlen/16){
    // printf("%i \n",index);
        AES_128(m, mlen, keys,index);

    }
}

void AES128Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys){
    aesBlock *mCuda;
    unsigned int *keysCuda;
    // printf("%x \n",keys[1167]);

    int sizeMessage = (mlen/16)*sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);

    dim3 nb( (unsigned int) ceil((double) mlen/16.0) ) ; 
    dim3 nt(2); 

    Encrypt<<<nb, nt>>>(mCuda, mlen,keysCuda);
    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);

    hipFree(mCuda); hipFree(keysCuda);  
}


__global__ void OCB128EncryptRandomAcces(aesBlock *m,aesBlock *delta, unsigned long long mlen, unsigned long long deltalen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    // int index2 = blockDim.x*blockIdx.x;
    if( index<mlen/16){
    // if( index==0){

        unsigned long long deltaIndex = floor( (double) index/255);
        
        __syncthreads();
        unsigned int mBlock[4];

        unsigned int deltaBlock[4];
        
        for (int i = 0 ; i< 4 ; i++){
            mBlock[i]= m[index].block[i];
            deltaBlock[i]= delta[deltaIndex].block[i]+index;
        }
        // falta añadir el checksum y si los bloques no vienen completos
        // imprimiArregloCudaInt(4,m[index].block );

        OCBAESDelta2Rounds(deltaBlock, keys);

        XOR_128(m[index].block,deltaBlock);

        // imprimiArregloCudaInt(4,m[index].block );

        AES_128(m, mlen, keys,index);

        XOR_128(m[index].block,deltaBlock);

    }
}
void OCBRandomAccess(aesBlock *m,aesBlock *delta,const unsigned long long mlen, unsigned long long deltalen, unsigned int *keys){
    aesBlock *mCuda;
    aesBlock *deltaCuda;
    unsigned int *keysCuda;
    // printf("%x \n",keys[1167]);

    int sizeMessage = (mlen/16)*sizeof(class aesBlock);
    int sizeDelta = (deltalen)*sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);
    hipMalloc(&deltaCuda, sizeDelta);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(deltaCuda,delta,sizeDelta,hipMemcpyDefault);

    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);

    dim3 nb( (unsigned int) ceil((double) mlen/16.0) ) ; 
    dim3 nt(2); 
     

    OCB128EncryptRandomAcces<<<nb, nt>>>(mCuda,deltaCuda, mlen, deltalen,keysCuda);
    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);

    hipFree(mCuda); hipFree(keysCuda);  
}

void getDelta(const unsigned int nonce[4],const unsigned long long mlen, aesBlock* delta,unsigned int *keys,unsigned long long deltalen ){
    
    for(int i = 0; i<deltalen; i++){
        
        for (int j = 0; j<4;j++){
            if(j==3){
                delta[i].block[j]= nonce[j]+i;
            }
            else{
                delta[i].block[j]= nonce[j];
            }

        }
    }
    AES128Encrypt(delta, deltalen*16, keys);


}

int main(int argc, char **argv) {
  

        // 0x2b,0x28,0xab,0x09,
        // 0x7e,0xae,0xf7,0xcf,
        // 0x15,0xd2,0x15,0x4f,
        // 0x16,0xa6,0x88,0x3c

    const unsigned int k[4] ={ 
        0x2b7e1516,
        0x28aed2a6,
        0xabf71588,
        0x09cf4f3c
    };
        // 0x32,0x88,0x31,0xe0,
        // 0x43,0x5a,0x31,0x37,
        // 0xf6,0x30,0x98,0x07,
        // 0xa8,0x8d,0xa2,0x34,
    const unsigned int nonce[4] ={ 
        0x3243f6a8,
        0X885a308d,
        0x313198a2,
        0xe0370734,
    };

        // 0x32,0x88,0x31,0xe0,
        // 0x43,0x5a,0x31,0x37,
        // 0xf6,0x30,0x98,0x07,
        // 0xa8,0x8d,0xa2,0x34,
    const unsigned int m[8] ={ 
        0x3243f6a8,
        0X885a308d,
        0x313198a2,
        0xe0370734,
        
        0x3243f6a8,
        0X885a308d,
        0x313198a2,
        0xe0370735
    };

    unsigned int result[64] ={ 
        0x32,0x88,0x31,0xe0,
        0x43,0x5a,0x31,0x37,
        0xf6,0x30,0x98,0x07,
        0xa8,0x8d,0xa2,0x34,

        0x32,0x88,0x31,0xe0,
        0x43,0x5a,0x31,0x37,
        0xf6,0x30,0x98,0x07,
        0xa8,0x8d,0xa2,0x34,

        0x32,0x88,0x31,0xe0,
        0x43,0x5a,0x31,0x37,
        0xf6,0x30,0x98,0x07,
        0xa8,0x8d,0xa2,0x34,

        0x32,0x88,0x31,0xe0,
        0x43,0x5a,0x31,0x37,
        0xf6,0x30,0x98,0x07,
        0xa8,0x8d,0xa2,0x34
    };
    unsigned long long mlen = 16;
    unsigned int keys[11][4];

    int numBlocks = mlen/16;
    aesBlock* aes_block_array;
    aes_block_array = new aesBlock [numBlocks];
    
    for(int i = 0; i<numBlocks; i++){
        for (int j = 0; j<4;j++){
            aes_block_array[i].block[j]=  m[(i*4)+j];
            // printf("0x%x\n",m[(i*4)+j] );
        }
    }
    ExpansionKeys128(k,1, keys );


    unsigned long long bloques = (unsigned long long) ceil( (double) mlen/16.0); //cada 4080 salta en 1 el delta
    unsigned long long deltalen = (unsigned long long) ceil( (double) bloques/255.0);
    aesBlock* delta;
    delta = new aesBlock [deltalen];
    getDelta(nonce , mlen, delta, &keys[0][0],deltalen);
    // imprimiArreglo(4,delta[0].block);
    OCBRandomAccess(aes_block_array, delta, mlen, deltalen, &keys[0][0]);

    // cout<<"delta "<<deltalen <<endl;
    // imprimiArreglo(4,delta[0].block);
    // imprimiArreglo(4,delta[1].block);

    // AES128Encrypt(aes_block_array, mlen, &keys[0][0],result);
    cout<<endl;

    for(int i = 0; i<numBlocks; i++){
        imprimiArreglo(4,aes_block_array[i].block);
        cout<<endl;
    }
    // imprimiArreglo(16,aes_block_array[1].block);
    // cout<<endl;

    // imprimiArreglo(16,aes_block_array[2].block);
    // cout<<endl;

    // imprimiArreglo(16,aes_block_array[3].block);
    // cout<<endl;
    
    
    
    return 0;
}