#include "hip/hip_runtime.h"
#include<iostream>
#include <cstring>
#include <xmmintrin.h>              /* SSE instructions and _mm_malloc */
#include <emmintrin.h>              /* SSE2 instructions               */
#include <tmmintrin.h>              /* SSSE3 instructions              */
#include <hip/hip_cooperative_groups.h> // Primary header is compatible with pre-C++11, collective algorithm headers require C++11

#define gpuchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#include<math.h>
using namespace std;
namespace cg = cooperative_groups;
#define NumberBlocks 32
#define NumnerThreads 256
#define L_TABLE_SZ 64;
#define OCB_TAG_LEN 16;
#define zero_block()          _mm_setzero_si128()
#define xor_block(x,y)        _mm_xor_si128(x,y)
#define unequal_blocks(x,y) \
    					   (_mm_movemask_epi8(_mm_cmpeq_epi8(x,y)) != 0xffff)
typedef __m128i block;
typedef struct { unsigned int keys[11][4]; int round; } AES_KEY;

struct _ae_ctx {
    AES_KEY decrypt_key;
    AES_KEY encrypt_key;
    unsigned int tag_len = OCB_TAG_LEN;
} ;
typedef struct _ae_ctx ae_ctx;

class aesBlock
{
public:
    unsigned int block[4] ={0,};
};

class LBlock
{
public:
    unsigned int block[4] ={0};
};

unsigned char matrizCajaS[256]={
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};


void ExpansionKeys128( unsigned int *k,  AES_KEY *aeskeys, AES_KEY *aeskeys_decrypt );
void unsignedCharArrayTounsignedIntArray(const unsigned char *in,unsigned int *out, unsigned long long len, unsigned long long mlen2 );
void unsignedCharArrayTounsignedIntArray(unsigned char *in,unsigned int *out, unsigned long long len, unsigned long long mlen2 );
void imprimiArreglo(int tam, unsigned int *keys );
void imprimiArreglo(int tam, unsigned char *keys );
void imprimiArreglo(int tam, const unsigned char *keys );
void imprimiArreglo(int tam, AES_KEY *aeskeys, int posicion );
void AES128Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys);
void shiftRowsMatrix(unsigned int* in, int *shifttab);
unsigned char GF2Redution(unsigned short in );
unsigned char multiplicacionENGF2(int caso , unsigned short numero2);
void print_hex_string(unsigned char* buf, int len);
void copyAESBlockToMessage(aesBlock* encrypt, int numBlocks, unsigned int * m2);
void copyMessageToAESBlock(aesBlock* encrypt, int numBlocks, unsigned int * m2 );


ae_ctx* ae_allocate(void *misc)
{
	void *p;
	(void) misc;                     /* misc unused in this implementation */
    p = _mm_malloc(sizeof(ae_ctx),16);
	return (ae_ctx *)p;
}

int ae_init(ae_ctx *ctx, const unsigned char *k, int key_len, int nonce_len, int tag_len)
{
    if (nonce_len != 12)
    	return 0;//AE_NOT_SUPPORTED
    /* Initialize encryption & decryption keys */
    unsigned int key[4];
    unsignedCharArrayTounsignedIntArray(k,key,16,16);
    //Falta las llaves de decifrado
    ExpansionKeys128( key,&ctx->encrypt_key,&ctx->decrypt_key );

    /* Zero things that need zeroing */
    // ctx->cached_Top = ctx->ad_checksum = zero_block();
    // ctx->ad_blocks_processed = 0;

    /* Compute key-dependent values */
    // aesBlock *tmp_aes_block = new aesBlock[1];

    // tmp_aes_block[0].block[0] = 0x03020100;
    // tmp_aes_block[0].block[1] = 0x07060504;
    // tmp_aes_block[0].block[2] = 0x0b0a0908;
    // tmp_aes_block[0].block[3] = 0x0f0e0d0c;

    // imprimiArreglo(4, (unsigned int *)&tmp_aes_block[0].block);

    // AES128Encrypt(tmp_aes_block, 16, &ctx->encrypt_key.keys[0][0]);

    // imprimiArreglo(4,(unsigned int *)&tmp_aes_block[0].block);
    // exit(1);

    // tmp_aes_block[0].block[0] = 0x0;
    // tmp_aes_block[0].block[1] = 0x0;
    // tmp_aes_block[0].block[2] = 0x0;
    // tmp_aes_block[0].block[3] = 0x0;
    // AES128Encrypt(tmp_aes_block, 16, &ctx->encrypt_key.keys[0][0]);

    ctx->tag_len = tag_len;


    return 1; //AE_SUCCESS;
}
void print_hex_string(unsigned char* buf, int len)
{
    int i;

    if (len==0) { printf("<empty string>"); return; }
    if (len>=40) {
        for (i = 0; i < 10; i++)
             printf("%02x", *((unsigned char *)buf + i));
        printf(" ... ");
        for (i = len-10; i < len; i++)
             printf("%02x", *((unsigned char *)buf + i));
        printf(" [%d bytes]", len);
        return;
    }
    for (i = 0; i < len; i++)
        printf("%02x", *((unsigned char *)buf + i));
}
void imprimiArreglo(int tam, unsigned int *keys ){
    for (int i = 0; i<tam; i++){
        printf("%08x", keys[i] );
    }
    printf("\n");
}
void imprimiArreglo(int tam, unsigned char *keys ){
    for (int i = 0; i<tam; i++){
        printf("%02x", keys[i] );
    }
    printf("\n");
}
void imprimiArreglo(int tam, const unsigned char *keys ){
    for (int i = 0; i<tam; i++){
        printf("%02x", keys[i] );
    }
    printf("\n");
}
void imprimiArreglo(int tam, AES_KEY *aeskeys, int posicion ){
    for (int i = 0; i<tam; i++){
        printf("%08x", aeskeys->keys[posicion][i] );
    }
    printf("\n");

}

void shiftRowsMatrix(unsigned int* in, int *shifttab){

    unsigned char * temp;
    unsigned char h[16];
    temp = (unsigned char *) in;
    memcpy(h, temp, 16);

    for(int i = 0; i < 16; i++){
        temp[i] = h[shifttab[i]];
    }
}
void ExpansionKeys128( unsigned int *k,  AES_KEY *aeskeys, AES_KEY *aeskeys_decrypt ){

    // imprimiArreglo(16,(unsigned char *)&k[0]);
    // imprimiArreglo(4,k);

    unsigned char RotWordTemp[4];
    const unsigned int matrizRcon[10]={ 0x01000000, 0x02000000, 0x04000000, 0x08000000, 0x10000000, 0x20000000, 0x40000000, 0x80000000, 0x1b000000, 0x36000000};
    memcpy(aeskeys->keys[0], k, 16);

    for(int i = 0; i<10; i++){
        RotWordTemp[0]=aeskeys->keys[i][3]>>16;
        RotWordTemp[1]=aeskeys->keys[i][3]>>8;
        RotWordTemp[2]=aeskeys->keys[i][3];
        RotWordTemp[3]=aeskeys->keys[i][3]>>24;

        for(int j = 0;  j < 4; j++ ){
            RotWordTemp[j] = matrizCajaS[ (int) RotWordTemp[j] ];
        }
        int RotWord = 0;
        RotWord = RotWord ^ ( (int) RotWordTemp[0])<<24;
        RotWord = RotWord ^ ( (int) RotWordTemp[1])<<16;
        RotWord = RotWord ^ ( (int) RotWordTemp[2])<<8;
        RotWord = RotWord ^ ( (int) RotWordTemp[3]);

        aeskeys->keys[i+1][0] =  RotWord ^ aeskeys->keys[i][0];
        aeskeys->keys[i+1][0] = aeskeys->keys[i+1][0] ^ matrizRcon[i];
        for(int x = 1;  x < 4; x++ ){
            aeskeys->keys[i+1][x] =  aeskeys->keys[i+1][x-1] ^ aeskeys->keys[i][x];
        }
    }

    aeskeys_decrypt->keys[0][0]=aeskeys->keys[10][0];
    aeskeys_decrypt->keys[0][1]=aeskeys->keys[10][1];
    aeskeys_decrypt->keys[0][2]=aeskeys->keys[10][2];
    aeskeys_decrypt->keys[0][3]=aeskeys->keys[10][3];
    unsigned char h[16];
    unsigned char resultado[16];
    int IMC[16]= {
        0x0e, 0x0b, 0x0d, 0x09,
        0x09, 0x0e, 0x0b, 0x0d,
        0x0d, 0x09, 0x0e, 0x0b,
        0x0b, 0x0d, 0x09, 0x0e

    };
    for(int k = 9; k>0;k--){
        memcpy(h, aeskeys->keys[k], 16);
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                unsigned char temp = 0;
                for(int k = 0; k < 4; k++ ){
                    temp= multiplicacionENGF2(IMC[ (j*4)+k ] , h[ (i*4) +3-k ]) ^ temp;
                }
                resultado[ (i*4) +3-j ] = temp;
            }
        }
        unsigned int * temporal = (unsigned int *) resultado;
        aeskeys_decrypt->keys[10-k][0] = temporal[0];
        aeskeys_decrypt->keys[10-k][1] = temporal[1];
        aeskeys_decrypt->keys[10-k][2] = temporal[2];
        aeskeys_decrypt->keys[10-k][3] = temporal[3];
    }

    aeskeys_decrypt->keys[10][0]=aeskeys->keys[0][0];
    aeskeys_decrypt->keys[10][1]=aeskeys->keys[0][1];
    aeskeys_decrypt->keys[10][2]=aeskeys->keys[0][2];
    aeskeys_decrypt->keys[10][3]=aeskeys->keys[0][3];
    int shifttab[16]={
        3, 2, 1, 0,
        7, 6, 5, 4,
        11, 10, 9, 8,
        15, 14, 13, 12,
    };
    for(int i = 0; i<11; i++){

        shiftRowsMatrix(aeskeys->keys[i], shifttab);
        shiftRowsMatrix(aeskeys_decrypt->keys[i], shifttab);
    }
}

__device__ void imprimiArregloCuda(int tam,unsigned char *keys ){
    for (int i = 0; i<tam; i++){
        printf("%02x ", keys[i] );
    }
    printf("\n");

}
__device__ void imprimiArregloCudaInt(int tam,unsigned int *keys ){
    printf("----------------\n" );
    for (int i = 0; i<tam; i++){
        printf("%08x ", keys[i] );
    }
    printf("\n");
}
__device__ void XOR_128(unsigned int* A, unsigned int * B )
{
        A[0] =A[0] ^  B[0];
        A[1] =A[1] ^  B[1];
        A[2] =A[2] ^  B[2];
        A[3] =A[3] ^  B[3];
}
__device__ void XOR2_128(unsigned int* A, unsigned int * B,unsigned int * C )
{
	for (int i = 0; i < 4; i++) {
        A[i] =C[i] ^  B[i];
    }
}
__device__ void addRoundKey(unsigned int* in, unsigned int * keys, int round)
{

	for (int i = 0; i < 4; i++) {
        in[i] =in[i] ^  keys[(round*4) + i];
    }

}
__device__ void subBytes(unsigned int* in, unsigned char * matrizCajaS)
{
    unsigned char * temp;
    temp = (unsigned char *) in;
	for (int i = 0; i < 16; i++) {
        temp[i] = matrizCajaS[ (int) temp[i]];
    }
}

__device__ void shiftRows(unsigned int* in, int *shifttab){

    unsigned char * temp;
    unsigned char h[16];
    temp = (unsigned char *) in;
    memcpy(h, temp, 16);

    for(int i = 0; i < 16; i++){
        temp[i] = h[shifttab[i]];
    }
}

__device__ void shiftRows2(unsigned int* in,unsigned int* out, int *shifttab){

    unsigned char * temp;
    unsigned char h[16];
    temp = (unsigned char *) in;
    memcpy(h, temp, 16);

    for(int i = 0; i < 16; i++){
        out[i] = h[shifttab[i]];
    }



}
unsigned char GF2Redution(unsigned short in ){

    unsigned short temp = in ; // numero de prueba
    // temp = temp * 2;
    unsigned char temp1 = temp>>8; //queda FF
    unsigned char temp2 = temp1<<1; //queda FE
    unsigned char temp3 = temp1<<3; //queda F8
    unsigned char temp4 = temp1<<4; //queda F

    unsigned char temp5 = temp>>13;//queda 7
    unsigned char temp6 = temp5<<1;//queda E
    unsigned char temp7 = temp5<<3;//queda 38
    unsigned char temp8 = temp5<<4;//queda 70

    unsigned char temp9 = temp>>12;//queda F
    unsigned char temp10 = temp9<<1;//queda 1E
    unsigned char temp11 = temp9<<3;//queda 78
    unsigned char temp12 = temp9<<4;//queda F0

    in = temp ^ temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6 ^ temp7 ^ temp8 ^ temp9 ^ temp10 ^ temp11 ^ temp12;
    return in;
}

unsigned char multiplicacionENGF2(int caso , unsigned short numero2){
    switch (caso)
    {
    case 1:
        return numero2;
        break;
    case 2:

        numero2 = GF2Redution(numero2 * 2);

        return numero2 ;
        break;
    case 3:

        numero2 = GF2Redution((numero2*2)^numero2);

        return numero2;
        break;
    case 9:

        numero2 = GF2Redution((numero2*8)^numero2);

        return numero2;
        break;

    case 11:


        numero2 = GF2Redution((numero2*8)) ^ GF2Redution((numero2*2)^numero2) ;

        return numero2;
        break;
    case 13:


        numero2 = GF2Redution((numero2*8)) ^ GF2Redution((numero2*4)) ^ numero2;

        return numero2;
        break;
    case 14:


        numero2 = GF2Redution((numero2*8)) ^ GF2Redution((numero2*4)) ^ GF2Redution((numero2*2));

        return numero2;
        break;

    default:

        break;
    }
    return 0;

}

__device__ void subBytesMixColumns(unsigned int* in, int * T1, int * T2, int * T3, int * T4){
	unsigned char * temp;
    temp = (unsigned char *) in;
    for (int i = 0; i < 16; i=i+4){
        int tempT1 = T1[(int) temp[i]];
        int tempT2 = T2[(int) temp[i+1]];
        int tempT3 = T3[(int) temp[i+2]];
        int tempT4 = T4[(int) temp[i+3] ];

        int tempT5 = tempT1 ^ tempT2 ^ tempT3 ^ tempT4;
        in[i/4] = tempT5;
    }
}


__constant__ unsigned char matrizCajaSTemp[256]={
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
    };

__constant__    int T1Temp[256] ={
        0xa56363c6U, 0x847c7cf8U, 0x997777eeU, 0x8d7b7bf6U,
        0x0df2f2ffU, 0xbd6b6bd6U, 0xb16f6fdeU, 0x54c5c591U,
        0x50303060U, 0x03010102U, 0xa96767ceU, 0x7d2b2b56U,
        0x19fefee7U, 0x62d7d7b5U, 0xe6abab4dU, 0x9a7676ecU,
        0x45caca8fU, 0x9d82821fU, 0x40c9c989U, 0x877d7dfaU,
        0x15fafaefU, 0xeb5959b2U, 0xc947478eU, 0x0bf0f0fbU,
        0xecadad41U, 0x67d4d4b3U, 0xfda2a25fU, 0xeaafaf45U,
        0xbf9c9c23U, 0xf7a4a453U, 0x967272e4U, 0x5bc0c09bU,
        0xc2b7b775U, 0x1cfdfde1U, 0xae93933dU, 0x6a26264cU,
        0x5a36366cU, 0x413f3f7eU, 0x02f7f7f5U, 0x4fcccc83U,
        0x5c343468U, 0xf4a5a551U, 0x34e5e5d1U, 0x08f1f1f9U,
        0x937171e2U, 0x73d8d8abU, 0x53313162U, 0x3f15152aU,
        0x0c040408U, 0x52c7c795U, 0x65232346U, 0x5ec3c39dU,
        0x28181830U, 0xa1969637U, 0x0f05050aU, 0xb59a9a2fU,
        0x0907070eU, 0x36121224U, 0x9b80801bU, 0x3de2e2dfU,
        0x26ebebcdU, 0x6927274eU, 0xcdb2b27fU, 0x9f7575eaU,
        0x1b090912U, 0x9e83831dU, 0x742c2c58U, 0x2e1a1a34U,
        0x2d1b1b36U, 0xb26e6edcU, 0xee5a5ab4U, 0xfba0a05bU,
        0xf65252a4U, 0x4d3b3b76U, 0x61d6d6b7U, 0xceb3b37dU,
        0x7b292952U, 0x3ee3e3ddU, 0x712f2f5eU, 0x97848413U,
        0xf55353a6U, 0x68d1d1b9U, 0x00000000U, 0x2cededc1U,
        0x60202040U, 0x1ffcfce3U, 0xc8b1b179U, 0xed5b5bb6U,
        0xbe6a6ad4U, 0x46cbcb8dU, 0xd9bebe67U, 0x4b393972U,
        0xde4a4a94U, 0xd44c4c98U, 0xe85858b0U, 0x4acfcf85U,
        0x6bd0d0bbU, 0x2aefefc5U, 0xe5aaaa4fU, 0x16fbfbedU,
        0xc5434386U, 0xd74d4d9aU, 0x55333366U, 0x94858511U,
        0xcf45458aU, 0x10f9f9e9U, 0x06020204U, 0x817f7ffeU,
        0xf05050a0U, 0x443c3c78U, 0xba9f9f25U, 0xe3a8a84bU,
        0xf35151a2U, 0xfea3a35dU, 0xc0404080U, 0x8a8f8f05U,
        0xad92923fU, 0xbc9d9d21U, 0x48383870U, 0x04f5f5f1U,
        0xdfbcbc63U, 0xc1b6b677U, 0x75dadaafU, 0x63212142U,
        0x30101020U, 0x1affffe5U, 0x0ef3f3fdU, 0x6dd2d2bfU,
        0x4ccdcd81U, 0x140c0c18U, 0x35131326U, 0x2fececc3U,
        0xe15f5fbeU, 0xa2979735U, 0xcc444488U, 0x3917172eU,
        0x57c4c493U, 0xf2a7a755U, 0x827e7efcU, 0x473d3d7aU,
        0xac6464c8U, 0xe75d5dbaU, 0x2b191932U, 0x957373e6U,
        0xa06060c0U, 0x98818119U, 0xd14f4f9eU, 0x7fdcdca3U,
        0x66222244U, 0x7e2a2a54U, 0xab90903bU, 0x8388880bU,
        0xca46468cU, 0x29eeeec7U, 0xd3b8b86bU, 0x3c141428U,
        0x79dedea7U, 0xe25e5ebcU, 0x1d0b0b16U, 0x76dbdbadU,
        0x3be0e0dbU, 0x56323264U, 0x4e3a3a74U, 0x1e0a0a14U,
        0xdb494992U, 0x0a06060cU, 0x6c242448U, 0xe45c5cb8U,
        0x5dc2c29fU, 0x6ed3d3bdU, 0xefacac43U, 0xa66262c4U,
        0xa8919139U, 0xa4959531U, 0x37e4e4d3U, 0x8b7979f2U,
        0x32e7e7d5U, 0x43c8c88bU, 0x5937376eU, 0xb76d6ddaU,
        0x8c8d8d01U, 0x64d5d5b1U, 0xd24e4e9cU, 0xe0a9a949U,
        0xb46c6cd8U, 0xfa5656acU, 0x07f4f4f3U, 0x25eaeacfU,
        0xaf6565caU, 0x8e7a7af4U, 0xe9aeae47U, 0x18080810U,
        0xd5baba6fU, 0x887878f0U, 0x6f25254aU, 0x722e2e5cU,
        0x241c1c38U, 0xf1a6a657U, 0xc7b4b473U, 0x51c6c697U,
        0x23e8e8cbU, 0x7cdddda1U, 0x9c7474e8U, 0x211f1f3eU,
        0xdd4b4b96U, 0xdcbdbd61U, 0x868b8b0dU, 0x858a8a0fU,
        0x907070e0U, 0x423e3e7cU, 0xc4b5b571U, 0xaa6666ccU,
        0xd8484890U, 0x05030306U, 0x01f6f6f7U, 0x120e0e1cU,
        0xa36161c2U, 0x5f35356aU, 0xf95757aeU, 0xd0b9b969U,
        0x91868617U, 0x58c1c199U, 0x271d1d3aU, 0xb99e9e27U,
        0x38e1e1d9U, 0x13f8f8ebU, 0xb398982bU, 0x33111122U,
        0xbb6969d2U, 0x70d9d9a9U, 0x898e8e07U, 0xa7949433U,
        0xb69b9b2dU, 0x221e1e3cU, 0x92878715U, 0x20e9e9c9U,
        0x49cece87U, 0xff5555aaU, 0x78282850U, 0x7adfdfa5U,
        0x8f8c8c03U, 0xf8a1a159U, 0x80898909U, 0x170d0d1aU,
        0xdabfbf65U, 0x31e6e6d7U, 0xc6424284U, 0xb86868d0U,
        0xc3414182U, 0xb0999929U, 0x772d2d5aU, 0x110f0f1eU,
        0xcbb0b07bU, 0xfc5454a8U, 0xd6bbbb6dU, 0x3a16162cU,


    };

__constant__    int T2Temp[256] = {



        0x6363c6a5U, 0x7c7cf884U, 0x7777ee99U, 0x7b7bf68dU,
        0xf2f2ff0dU, 0x6b6bd6bdU, 0x6f6fdeb1U, 0xc5c59154U,
        0x30306050U, 0x01010203U, 0x6767cea9U, 0x2b2b567dU,
        0xfefee719U, 0xd7d7b562U, 0xabab4de6U, 0x7676ec9aU,
        0xcaca8f45U, 0x82821f9dU, 0xc9c98940U, 0x7d7dfa87U,
        0xfafaef15U, 0x5959b2ebU, 0x47478ec9U, 0xf0f0fb0bU,
        0xadad41ecU, 0xd4d4b367U, 0xa2a25ffdU, 0xafaf45eaU,
        0x9c9c23bfU, 0xa4a453f7U, 0x7272e496U, 0xc0c09b5bU,
        0xb7b775c2U, 0xfdfde11cU, 0x93933daeU, 0x26264c6aU,
        0x36366c5aU, 0x3f3f7e41U, 0xf7f7f502U, 0xcccc834fU,
        0x3434685cU, 0xa5a551f4U, 0xe5e5d134U, 0xf1f1f908U,
        0x7171e293U, 0xd8d8ab73U, 0x31316253U, 0x15152a3fU,
        0x0404080cU, 0xc7c79552U, 0x23234665U, 0xc3c39d5eU,
        0x18183028U, 0x969637a1U, 0x05050a0fU, 0x9a9a2fb5U,
        0x07070e09U, 0x12122436U, 0x80801b9bU, 0xe2e2df3dU,
        0xebebcd26U, 0x27274e69U, 0xb2b27fcdU, 0x7575ea9fU,
        0x0909121bU, 0x83831d9eU, 0x2c2c5874U, 0x1a1a342eU,
        0x1b1b362dU, 0x6e6edcb2U, 0x5a5ab4eeU, 0xa0a05bfbU,
        0x5252a4f6U, 0x3b3b764dU, 0xd6d6b761U, 0xb3b37dceU,
        0x2929527bU, 0xe3e3dd3eU, 0x2f2f5e71U, 0x84841397U,
        0x5353a6f5U, 0xd1d1b968U, 0x00000000U, 0xededc12cU,
        0x20204060U, 0xfcfce31fU, 0xb1b179c8U, 0x5b5bb6edU,
        0x6a6ad4beU, 0xcbcb8d46U, 0xbebe67d9U, 0x3939724bU,
        0x4a4a94deU, 0x4c4c98d4U, 0x5858b0e8U, 0xcfcf854aU,
        0xd0d0bb6bU, 0xefefc52aU, 0xaaaa4fe5U, 0xfbfbed16U,
        0x434386c5U, 0x4d4d9ad7U, 0x33336655U, 0x85851194U,
        0x45458acfU, 0xf9f9e910U, 0x02020406U, 0x7f7ffe81U,
        0x5050a0f0U, 0x3c3c7844U, 0x9f9f25baU, 0xa8a84be3U,
        0x5151a2f3U, 0xa3a35dfeU, 0x404080c0U, 0x8f8f058aU,
        0x92923fadU, 0x9d9d21bcU, 0x38387048U, 0xf5f5f104U,
        0xbcbc63dfU, 0xb6b677c1U, 0xdadaaf75U, 0x21214263U,
        0x10102030U, 0xffffe51aU, 0xf3f3fd0eU, 0xd2d2bf6dU,
        0xcdcd814cU, 0x0c0c1814U, 0x13132635U, 0xececc32fU,
        0x5f5fbee1U, 0x979735a2U, 0x444488ccU, 0x17172e39U,
        0xc4c49357U, 0xa7a755f2U, 0x7e7efc82U, 0x3d3d7a47U,
        0x6464c8acU, 0x5d5dbae7U, 0x1919322bU, 0x7373e695U,
        0x6060c0a0U, 0x81811998U, 0x4f4f9ed1U, 0xdcdca37fU,
        0x22224466U, 0x2a2a547eU, 0x90903babU, 0x88880b83U,
        0x46468ccaU, 0xeeeec729U, 0xb8b86bd3U, 0x1414283cU,
        0xdedea779U, 0x5e5ebce2U, 0x0b0b161dU, 0xdbdbad76U,
        0xe0e0db3bU, 0x32326456U, 0x3a3a744eU, 0x0a0a141eU,
        0x494992dbU, 0x06060c0aU, 0x2424486cU, 0x5c5cb8e4U,
        0xc2c29f5dU, 0xd3d3bd6eU, 0xacac43efU, 0x6262c4a6U,
        0x919139a8U, 0x959531a4U, 0xe4e4d337U, 0x7979f28bU,
        0xe7e7d532U, 0xc8c88b43U, 0x37376e59U, 0x6d6ddab7U,
        0x8d8d018cU, 0xd5d5b164U, 0x4e4e9cd2U, 0xa9a949e0U,
        0x6c6cd8b4U, 0x5656acfaU, 0xf4f4f307U, 0xeaeacf25U,
        0x6565caafU, 0x7a7af48eU, 0xaeae47e9U, 0x08081018U,
        0xbaba6fd5U, 0x7878f088U, 0x25254a6fU, 0x2e2e5c72U,
        0x1c1c3824U, 0xa6a657f1U, 0xb4b473c7U, 0xc6c69751U,
        0xe8e8cb23U, 0xdddda17cU, 0x7474e89cU, 0x1f1f3e21U,
        0x4b4b96ddU, 0xbdbd61dcU, 0x8b8b0d86U, 0x8a8a0f85U,
        0x7070e090U, 0x3e3e7c42U, 0xb5b571c4U, 0x6666ccaaU,
        0x484890d8U, 0x03030605U, 0xf6f6f701U, 0x0e0e1c12U,
        0x6161c2a3U, 0x35356a5fU, 0x5757aef9U, 0xb9b969d0U,
        0x86861791U, 0xc1c19958U, 0x1d1d3a27U, 0x9e9e27b9U,
        0xe1e1d938U, 0xf8f8eb13U, 0x98982bb3U, 0x11112233U,
        0x6969d2bbU, 0xd9d9a970U, 0x8e8e0789U, 0x949433a7U,
        0x9b9b2db6U, 0x1e1e3c22U, 0x87871592U, 0xe9e9c920U,
        0xcece8749U, 0x5555aaffU, 0x28285078U, 0xdfdfa57aU,
        0x8c8c038fU, 0xa1a159f8U, 0x89890980U, 0x0d0d1a17U,
        0xbfbf65daU, 0xe6e6d731U, 0x424284c6U, 0x6868d0b8U,
        0x414182c3U, 0x999929b0U, 0x2d2d5a77U, 0x0f0f1e11U,
        0xb0b07bcbU, 0x5454a8fcU, 0xbbbb6dd6U, 0x16162c3aU,



      } ;

__constant__    int T3Temp[256] = {

        0x63c6a563U, 0x7cf8847cU, 0x77ee9977U, 0x7bf68d7bU,
        0xf2ff0df2U, 0x6bd6bd6bU, 0x6fdeb16fU, 0xc59154c5U,
        0x30605030U, 0x01020301U, 0x67cea967U, 0x2b567d2bU,
        0xfee719feU, 0xd7b562d7U, 0xab4de6abU, 0x76ec9a76U,
        0xca8f45caU, 0x821f9d82U, 0xc98940c9U, 0x7dfa877dU,
        0xfaef15faU, 0x59b2eb59U, 0x478ec947U, 0xf0fb0bf0U,
        0xad41ecadU, 0xd4b367d4U, 0xa25ffda2U, 0xaf45eaafU,
        0x9c23bf9cU, 0xa453f7a4U, 0x72e49672U, 0xc09b5bc0U,
        0xb775c2b7U, 0xfde11cfdU, 0x933dae93U, 0x264c6a26U,
        0x366c5a36U, 0x3f7e413fU, 0xf7f502f7U, 0xcc834fccU,
        0x34685c34U, 0xa551f4a5U, 0xe5d134e5U, 0xf1f908f1U,
        0x71e29371U, 0xd8ab73d8U, 0x31625331U, 0x152a3f15U,
        0x04080c04U, 0xc79552c7U, 0x23466523U, 0xc39d5ec3U,
        0x18302818U, 0x9637a196U, 0x050a0f05U, 0x9a2fb59aU,
        0x070e0907U, 0x12243612U, 0x801b9b80U, 0xe2df3de2U,
        0xebcd26ebU, 0x274e6927U, 0xb27fcdb2U, 0x75ea9f75U,
        0x09121b09U, 0x831d9e83U, 0x2c58742cU, 0x1a342e1aU,
        0x1b362d1bU, 0x6edcb26eU, 0x5ab4ee5aU, 0xa05bfba0U,
        0x52a4f652U, 0x3b764d3bU, 0xd6b761d6U, 0xb37dceb3U,
        0x29527b29U, 0xe3dd3ee3U, 0x2f5e712fU, 0x84139784U,
        0x53a6f553U, 0xd1b968d1U, 0x00000000U, 0xedc12cedU,
        0x20406020U, 0xfce31ffcU, 0xb179c8b1U, 0x5bb6ed5bU,
        0x6ad4be6aU, 0xcb8d46cbU, 0xbe67d9beU, 0x39724b39U,
        0x4a94de4aU, 0x4c98d44cU, 0x58b0e858U, 0xcf854acfU,
        0xd0bb6bd0U, 0xefc52aefU, 0xaa4fe5aaU, 0xfbed16fbU,
        0x4386c543U, 0x4d9ad74dU, 0x33665533U, 0x85119485U,
        0x458acf45U, 0xf9e910f9U, 0x02040602U, 0x7ffe817fU,
        0x50a0f050U, 0x3c78443cU, 0x9f25ba9fU, 0xa84be3a8U,
        0x51a2f351U, 0xa35dfea3U, 0x4080c040U, 0x8f058a8fU,
        0x923fad92U, 0x9d21bc9dU, 0x38704838U, 0xf5f104f5U,
        0xbc63dfbcU, 0xb677c1b6U, 0xdaaf75daU, 0x21426321U,
        0x10203010U, 0xffe51affU, 0xf3fd0ef3U, 0xd2bf6dd2U,
        0xcd814ccdU, 0x0c18140cU, 0x13263513U, 0xecc32fecU,
        0x5fbee15fU, 0x9735a297U, 0x4488cc44U, 0x172e3917U,
        0xc49357c4U, 0xa755f2a7U, 0x7efc827eU, 0x3d7a473dU,
        0x64c8ac64U, 0x5dbae75dU, 0x19322b19U, 0x73e69573U,
        0x60c0a060U, 0x81199881U, 0x4f9ed14fU, 0xdca37fdcU,
        0x22446622U, 0x2a547e2aU, 0x903bab90U, 0x880b8388U,
        0x468cca46U, 0xeec729eeU, 0xb86bd3b8U, 0x14283c14U,
        0xdea779deU, 0x5ebce25eU, 0x0b161d0bU, 0xdbad76dbU,
        0xe0db3be0U, 0x32645632U, 0x3a744e3aU, 0x0a141e0aU,
        0x4992db49U, 0x060c0a06U, 0x24486c24U, 0x5cb8e45cU,
        0xc29f5dc2U, 0xd3bd6ed3U, 0xac43efacU, 0x62c4a662U,
        0x9139a891U, 0x9531a495U, 0xe4d337e4U, 0x79f28b79U,
        0xe7d532e7U, 0xc88b43c8U, 0x376e5937U, 0x6ddab76dU,
        0x8d018c8dU, 0xd5b164d5U, 0x4e9cd24eU, 0xa949e0a9U,
        0x6cd8b46cU, 0x56acfa56U, 0xf4f307f4U, 0xeacf25eaU,
        0x65caaf65U, 0x7af48e7aU, 0xae47e9aeU, 0x08101808U,
        0xba6fd5baU, 0x78f08878U, 0x254a6f25U, 0x2e5c722eU,
        0x1c38241cU, 0xa657f1a6U, 0xb473c7b4U, 0xc69751c6U,
        0xe8cb23e8U, 0xdda17cddU, 0x74e89c74U, 0x1f3e211fU,
        0x4b96dd4bU, 0xbd61dcbdU, 0x8b0d868bU, 0x8a0f858aU,
        0x70e09070U, 0x3e7c423eU, 0xb571c4b5U, 0x66ccaa66U,
        0x4890d848U, 0x03060503U, 0xf6f701f6U, 0x0e1c120eU,
        0x61c2a361U, 0x356a5f35U, 0x57aef957U, 0xb969d0b9U,
        0x86179186U, 0xc19958c1U, 0x1d3a271dU, 0x9e27b99eU,
        0xe1d938e1U, 0xf8eb13f8U, 0x982bb398U, 0x11223311U,
        0x69d2bb69U, 0xd9a970d9U, 0x8e07898eU, 0x9433a794U,
        0x9b2db69bU, 0x1e3c221eU, 0x87159287U, 0xe9c920e9U,
        0xce8749ceU, 0x55aaff55U, 0x28507828U, 0xdfa57adfU,
        0x8c038f8cU, 0xa159f8a1U, 0x89098089U, 0x0d1a170dU,
        0xbf65dabfU, 0xe6d731e6U, 0x4284c642U, 0x68d0b868U,
        0x4182c341U, 0x9929b099U, 0x2d5a772dU, 0x0f1e110fU,
        0xb07bcbb0U, 0x54a8fc54U, 0xbb6dd6bbU, 0x162c3a16U,
    };

__constant__    int T4Temp[256] = {
        0xc6a56363U, 0xf8847c7cU, 0xee997777U, 0xf68d7b7bU,
        0xff0df2f2U, 0xd6bd6b6bU, 0xdeb16f6fU, 0x9154c5c5U,
        0x60503030U, 0x02030101U, 0xcea96767U, 0x567d2b2bU,
        0xe719fefeU, 0xb562d7d7U, 0x4de6ababU, 0xec9a7676U,
        0x8f45cacaU, 0x1f9d8282U, 0x8940c9c9U, 0xfa877d7dU,
        0xef15fafaU, 0xb2eb5959U, 0x8ec94747U, 0xfb0bf0f0U,
        0x41ecadadU, 0xb367d4d4U, 0x5ffda2a2U, 0x45eaafafU,
        0x23bf9c9cU, 0x53f7a4a4U, 0xe4967272U, 0x9b5bc0c0U,
        0x75c2b7b7U, 0xe11cfdfdU, 0x3dae9393U, 0x4c6a2626U,
        0x6c5a3636U, 0x7e413f3fU, 0xf502f7f7U, 0x834fccccU,
        0x685c3434U, 0x51f4a5a5U, 0xd134e5e5U, 0xf908f1f1U,
        0xe2937171U, 0xab73d8d8U, 0x62533131U, 0x2a3f1515U,
        0x080c0404U, 0x9552c7c7U, 0x46652323U, 0x9d5ec3c3U,
        0x30281818U, 0x37a19696U, 0x0a0f0505U, 0x2fb59a9aU,
        0x0e090707U, 0x24361212U, 0x1b9b8080U, 0xdf3de2e2U,
        0xcd26ebebU, 0x4e692727U, 0x7fcdb2b2U, 0xea9f7575U,
        0x121b0909U, 0x1d9e8383U, 0x58742c2cU, 0x342e1a1aU,
        0x362d1b1bU, 0xdcb26e6eU, 0xb4ee5a5aU, 0x5bfba0a0U,
        0xa4f65252U, 0x764d3b3bU, 0xb761d6d6U, 0x7dceb3b3U,
        0x527b2929U, 0xdd3ee3e3U, 0x5e712f2fU, 0x13978484U,
        0xa6f55353U, 0xb968d1d1U, 0x00000000U, 0xc12cededU,
        0x40602020U, 0xe31ffcfcU, 0x79c8b1b1U, 0xb6ed5b5bU,
        0xd4be6a6aU, 0x8d46cbcbU, 0x67d9bebeU, 0x724b3939U,
        0x94de4a4aU, 0x98d44c4cU, 0xb0e85858U, 0x854acfcfU,
        0xbb6bd0d0U, 0xc52aefefU, 0x4fe5aaaaU, 0xed16fbfbU,
        0x86c54343U, 0x9ad74d4dU, 0x66553333U, 0x11948585U,
        0x8acf4545U, 0xe910f9f9U, 0x04060202U, 0xfe817f7fU,
        0xa0f05050U, 0x78443c3cU, 0x25ba9f9fU, 0x4be3a8a8U,
        0xa2f35151U, 0x5dfea3a3U, 0x80c04040U, 0x058a8f8fU,
        0x3fad9292U, 0x21bc9d9dU, 0x70483838U, 0xf104f5f5U,
        0x63dfbcbcU, 0x77c1b6b6U, 0xaf75dadaU, 0x42632121U,
        0x20301010U, 0xe51affffU, 0xfd0ef3f3U, 0xbf6dd2d2U,
        0x814ccdcdU, 0x18140c0cU, 0x26351313U, 0xc32fececU,
        0xbee15f5fU, 0x35a29797U, 0x88cc4444U, 0x2e391717U,
        0x9357c4c4U, 0x55f2a7a7U, 0xfc827e7eU, 0x7a473d3dU,
        0xc8ac6464U, 0xbae75d5dU, 0x322b1919U, 0xe6957373U,
        0xc0a06060U, 0x19988181U, 0x9ed14f4fU, 0xa37fdcdcU,
        0x44662222U, 0x547e2a2aU, 0x3bab9090U, 0x0b838888U,
        0x8cca4646U, 0xc729eeeeU, 0x6bd3b8b8U, 0x283c1414U,
        0xa779dedeU, 0xbce25e5eU, 0x161d0b0bU, 0xad76dbdbU,
        0xdb3be0e0U, 0x64563232U, 0x744e3a3aU, 0x141e0a0aU,
        0x92db4949U, 0x0c0a0606U, 0x486c2424U, 0xb8e45c5cU,
        0x9f5dc2c2U, 0xbd6ed3d3U, 0x43efacacU, 0xc4a66262U,
        0x39a89191U, 0x31a49595U, 0xd337e4e4U, 0xf28b7979U,
        0xd532e7e7U, 0x8b43c8c8U, 0x6e593737U, 0xdab76d6dU,
        0x018c8d8dU, 0xb164d5d5U, 0x9cd24e4eU, 0x49e0a9a9U,
        0xd8b46c6cU, 0xacfa5656U, 0xf307f4f4U, 0xcf25eaeaU,
        0xcaaf6565U, 0xf48e7a7aU, 0x47e9aeaeU, 0x10180808U,
        0x6fd5babaU, 0xf0887878U, 0x4a6f2525U, 0x5c722e2eU,
        0x38241c1cU, 0x57f1a6a6U, 0x73c7b4b4U, 0x9751c6c6U,
        0xcb23e8e8U, 0xa17cddddU, 0xe89c7474U, 0x3e211f1fU,
        0x96dd4b4bU, 0x61dcbdbdU, 0x0d868b8bU, 0x0f858a8aU,
        0xe0907070U, 0x7c423e3eU, 0x71c4b5b5U, 0xccaa6666U,
        0x90d84848U, 0x06050303U, 0xf701f6f6U, 0x1c120e0eU,
        0xc2a36161U, 0x6a5f3535U, 0xaef95757U, 0x69d0b9b9U,
        0x17918686U, 0x9958c1c1U, 0x3a271d1dU, 0x27b99e9eU,
        0xd938e1e1U, 0xeb13f8f8U, 0x2bb39898U, 0x22331111U,
        0xd2bb6969U, 0xa970d9d9U, 0x07898e8eU, 0x33a79494U,
        0x2db69b9bU, 0x3c221e1eU, 0x15928787U, 0xc920e9e9U,
        0x8749ceceU, 0xaaff5555U, 0x50782828U, 0xa57adfdfU,
        0x038f8c8cU, 0x59f8a1a1U, 0x09808989U, 0x1a170d0dU,
        0x65dabfbfU, 0xd731e6e6U, 0x84c64242U, 0xd0b86868U,
        0x82c34141U, 0x29b09999U, 0x5a772d2dU, 0x1e110f0fU,
        0x7bcbb0b0U, 0xa8fc5454U, 0x6dd6bbbbU, 0x2c3a1616U,

    };




__device__ int AES_init_decrypt( unsigned char  *matrizCajaS, int *T1, int *T2, int *T3, int *T4){

    int T1Temp[256] ={

        0x50a7f451U, 0x5365417eU, 0xc3a4171aU, 0x965e273aU,
        0xcb6bab3bU, 0xf1459d1fU, 0xab58faacU, 0x9303e34bU,
        0x55fa3020U, 0xf66d76adU, 0x9176cc88U, 0x254c02f5U,
        0xfcd7e54fU, 0xd7cb2ac5U, 0x80443526U, 0x8fa362b5U,
        0x495ab1deU, 0x671bba25U, 0x980eea45U, 0xe1c0fe5dU,
        0x02752fc3U, 0x12f04c81U, 0xa397468dU, 0xc6f9d36bU,
        0xe75f8f03U, 0x959c9215U, 0xeb7a6dbfU, 0xda595295U,
        0x2d83bed4U, 0xd3217458U, 0x2969e049U, 0x44c8c98eU,
        0x6a89c275U, 0x78798ef4U, 0x6b3e5899U, 0xdd71b927U,
        0xb64fe1beU, 0x17ad88f0U, 0x66ac20c9U, 0xb43ace7dU,
        0x184adf63U, 0x82311ae5U, 0x60335197U, 0x457f5362U,
        0xe07764b1U, 0x84ae6bbbU, 0x1ca081feU, 0x942b08f9U,
        0x58684870U, 0x19fd458fU, 0x876cde94U, 0xb7f87b52U,
        0x23d373abU, 0xe2024b72U, 0x578f1fe3U, 0x2aab5566U,
        0x0728ebb2U, 0x03c2b52fU, 0x9a7bc586U, 0xa50837d3U,
        0xf2872830U, 0xb2a5bf23U, 0xba6a0302U, 0x5c8216edU,
        0x2b1ccf8aU, 0x92b479a7U, 0xf0f207f3U, 0xa1e2694eU,
        0xcdf4da65U, 0xd5be0506U, 0x1f6234d1U, 0x8afea6c4U,
        0x9d532e34U, 0xa055f3a2U, 0x32e18a05U, 0x75ebf6a4U,
        0x39ec830bU, 0xaaef6040U, 0x069f715eU, 0x51106ebdU,
        0xf98a213eU, 0x3d06dd96U, 0xae053eddU, 0x46bde64dU,
        0xb58d5491U, 0x055dc471U, 0x6fd40604U, 0xff155060U,
        0x24fb9819U, 0x97e9bdd6U, 0xcc434089U, 0x779ed967U,
        0xbd42e8b0U, 0x888b8907U, 0x385b19e7U, 0xdbeec879U,
        0x470a7ca1U, 0xe90f427cU, 0xc91e84f8U, 0x00000000U,
        0x83868009U, 0x48ed2b32U, 0xac70111eU, 0x4e725a6cU,
        0xfbff0efdU, 0x5638850fU, 0x1ed5ae3dU, 0x27392d36U,
        0x64d90f0aU, 0x21a65c68U, 0xd1545b9bU, 0x3a2e3624U,
        0xb1670a0cU, 0x0fe75793U, 0xd296eeb4U, 0x9e919b1bU,
        0x4fc5c080U, 0xa220dc61U, 0x694b775aU, 0x161a121cU,
        0x0aba93e2U, 0xe52aa0c0U, 0x43e0223cU, 0x1d171b12U,
        0x0b0d090eU, 0xadc78bf2U, 0xb9a8b62dU, 0xc8a91e14U,
        0x8519f157U, 0x4c0775afU, 0xbbdd99eeU, 0xfd607fa3U,
        0x9f2601f7U, 0xbcf5725cU, 0xc53b6644U, 0x347efb5bU,
        0x7629438bU, 0xdcc623cbU, 0x68fcedb6U, 0x63f1e4b8U,
        0xcadc31d7U, 0x10856342U, 0x40229713U, 0x2011c684U,
        0x7d244a85U, 0xf83dbbd2U, 0x1132f9aeU, 0x6da129c7U,
        0x4b2f9e1dU, 0xf330b2dcU, 0xec52860dU, 0xd0e3c177U,
        0x6c16b32bU, 0x99b970a9U, 0xfa489411U, 0x2264e947U,
        0xc48cfca8U, 0x1a3ff0a0U, 0xd82c7d56U, 0xef903322U,
        0xc74e4987U, 0xc1d138d9U, 0xfea2ca8cU, 0x360bd498U,
        0xcf81f5a6U, 0x28de7aa5U, 0x268eb7daU, 0xa4bfad3fU,
        0xe49d3a2cU, 0x0d927850U, 0x9bcc5f6aU, 0x62467e54U,
        0xc2138df6U, 0xe8b8d890U, 0x5ef7392eU, 0xf5afc382U,
        0xbe805d9fU, 0x7c93d069U, 0xa92dd56fU, 0xb31225cfU,
        0x3b99acc8U, 0xa77d1810U, 0x6e639ce8U, 0x7bbb3bdbU,
        0x097826cdU, 0xf418596eU, 0x01b79aecU, 0xa89a4f83U,
        0x656e95e6U, 0x7ee6ffaaU, 0x08cfbc21U, 0xe6e815efU,
        0xd99be7baU, 0xce366f4aU, 0xd4099feaU, 0xd67cb029U,
        0xafb2a431U, 0x31233f2aU, 0x3094a5c6U, 0xc066a235U,
        0x37bc4e74U, 0xa6ca82fcU, 0xb0d090e0U, 0x15d8a733U,
        0x4a9804f1U, 0xf7daec41U, 0x0e50cd7fU, 0x2ff69117U,
        0x8dd64d76U, 0x4db0ef43U, 0x544daaccU, 0xdf0496e4U,
        0xe3b5d19eU, 0x1b886a4cU, 0xb81f2cc1U, 0x7f516546U,
        0x04ea5e9dU, 0x5d358c01U, 0x737487faU, 0x2e410bfbU,
        0x5a1d67b3U, 0x52d2db92U, 0x335610e9U, 0x1347d66dU,
        0x8c61d79aU, 0x7a0ca137U, 0x8e14f859U, 0x893c13ebU,
        0xee27a9ceU, 0x35c961b7U, 0xede51ce1U, 0x3cb1477aU,
        0x59dfd29cU, 0x3f73f255U, 0x79ce1418U, 0xbf37c773U,
        0xeacdf753U, 0x5baafd5fU, 0x146f3ddfU, 0x86db4478U,
        0x81f3afcaU, 0x3ec468b9U, 0x2c342438U, 0x5f40a3c2U,
        0x72c31d16U, 0x0c25e2bcU, 0x8b493c28U, 0x41950dffU,
        0x7101a839U, 0xdeb30c08U, 0x9ce4b4d8U, 0x90c15664U,
        0x6184cb7bU, 0x70b632d5U, 0x745c6c48U, 0x4257b8d0U,

        };

    int T2Temp[256] = {

        0xa7f45150U, 0x65417e53U, 0xa4171ac3U, 0x5e273a96U,
        0x6bab3bcbU, 0x459d1ff1U, 0x58faacabU, 0x03e34b93U,
        0xfa302055U, 0x6d76adf6U, 0x76cc8891U, 0x4c02f525U,
        0xd7e54ffcU, 0xcb2ac5d7U, 0x44352680U, 0xa362b58fU,
        0x5ab1de49U, 0x1bba2567U, 0x0eea4598U, 0xc0fe5de1U,
        0x752fc302U, 0xf04c8112U, 0x97468da3U, 0xf9d36bc6U,
        0x5f8f03e7U, 0x9c921595U, 0x7a6dbfebU, 0x595295daU,
        0x83bed42dU, 0x217458d3U, 0x69e04929U, 0xc8c98e44U,
        0x89c2756aU, 0x798ef478U, 0x3e58996bU, 0x71b927ddU,
        0x4fe1beb6U, 0xad88f017U, 0xac20c966U, 0x3ace7db4U,
        0x4adf6318U, 0x311ae582U, 0x33519760U, 0x7f536245U,
        0x7764b1e0U, 0xae6bbb84U, 0xa081fe1cU, 0x2b08f994U,
        0x68487058U, 0xfd458f19U, 0x6cde9487U, 0xf87b52b7U,
        0xd373ab23U, 0x024b72e2U, 0x8f1fe357U, 0xab55662aU,
        0x28ebb207U, 0xc2b52f03U, 0x7bc5869aU, 0x0837d3a5U,
        0x872830f2U, 0xa5bf23b2U, 0x6a0302baU, 0x8216ed5cU,
        0x1ccf8a2bU, 0xb479a792U, 0xf207f3f0U, 0xe2694ea1U,
        0xf4da65cdU, 0xbe0506d5U, 0x6234d11fU, 0xfea6c48aU,
        0x532e349dU, 0x55f3a2a0U, 0xe18a0532U, 0xebf6a475U,
        0xec830b39U, 0xef6040aaU, 0x9f715e06U, 0x106ebd51U,
        0x8a213ef9U, 0x06dd963dU, 0x053eddaeU, 0xbde64d46U,
        0x8d5491b5U, 0x5dc47105U, 0xd406046fU, 0x155060ffU,
        0xfb981924U, 0xe9bdd697U, 0x434089ccU, 0x9ed96777U,
        0x42e8b0bdU, 0x8b890788U, 0x5b19e738U, 0xeec879dbU,
        0x0a7ca147U, 0x0f427ce9U, 0x1e84f8c9U, 0x00000000U,
        0x86800983U, 0xed2b3248U, 0x70111eacU, 0x725a6c4eU,
        0xff0efdfbU, 0x38850f56U, 0xd5ae3d1eU, 0x392d3627U,
        0xd90f0a64U, 0xa65c6821U, 0x545b9bd1U, 0x2e36243aU,
        0x670a0cb1U, 0xe757930fU, 0x96eeb4d2U, 0x919b1b9eU,
        0xc5c0804fU, 0x20dc61a2U, 0x4b775a69U, 0x1a121c16U,
        0xba93e20aU, 0x2aa0c0e5U, 0xe0223c43U, 0x171b121dU,
        0x0d090e0bU, 0xc78bf2adU, 0xa8b62db9U, 0xa91e14c8U,
        0x19f15785U, 0x0775af4cU, 0xdd99eebbU, 0x607fa3fdU,
        0x2601f79fU, 0xf5725cbcU, 0x3b6644c5U, 0x7efb5b34U,
        0x29438b76U, 0xc623cbdcU, 0xfcedb668U, 0xf1e4b863U,
        0xdc31d7caU, 0x85634210U, 0x22971340U, 0x11c68420U,
        0x244a857dU, 0x3dbbd2f8U, 0x32f9ae11U, 0xa129c76dU,
        0x2f9e1d4bU, 0x30b2dcf3U, 0x52860decU, 0xe3c177d0U,
        0x16b32b6cU, 0xb970a999U, 0x489411faU, 0x64e94722U,
        0x8cfca8c4U, 0x3ff0a01aU, 0x2c7d56d8U, 0x903322efU,
        0x4e4987c7U, 0xd138d9c1U, 0xa2ca8cfeU, 0x0bd49836U,
        0x81f5a6cfU, 0xde7aa528U, 0x8eb7da26U, 0xbfad3fa4U,
        0x9d3a2ce4U, 0x9278500dU, 0xcc5f6a9bU, 0x467e5462U,
        0x138df6c2U, 0xb8d890e8U, 0xf7392e5eU, 0xafc382f5U,
        0x805d9fbeU, 0x93d0697cU, 0x2dd56fa9U, 0x1225cfb3U,
        0x99acc83bU, 0x7d1810a7U, 0x639ce86eU, 0xbb3bdb7bU,
        0x7826cd09U, 0x18596ef4U, 0xb79aec01U, 0x9a4f83a8U,
        0x6e95e665U, 0xe6ffaa7eU, 0xcfbc2108U, 0xe815efe6U,
        0x9be7bad9U, 0x366f4aceU, 0x099fead4U, 0x7cb029d6U,
        0xb2a431afU, 0x233f2a31U, 0x94a5c630U, 0x66a235c0U,
        0xbc4e7437U, 0xca82fca6U, 0xd090e0b0U, 0xd8a73315U,
        0x9804f14aU, 0xdaec41f7U, 0x50cd7f0eU, 0xf691172fU,
        0xd64d768dU, 0xb0ef434dU, 0x4daacc54U, 0x0496e4dfU,
        0xb5d19ee3U, 0x886a4c1bU, 0x1f2cc1b8U, 0x5165467fU,
        0xea5e9d04U, 0x358c015dU, 0x7487fa73U, 0x410bfb2eU,
        0x1d67b35aU, 0xd2db9252U, 0x5610e933U, 0x47d66d13U,
        0x61d79a8cU, 0x0ca1377aU, 0x14f8598eU, 0x3c13eb89U,
        0x27a9ceeeU, 0xc961b735U, 0xe51ce1edU, 0xb1477a3cU,
        0xdfd29c59U, 0x73f2553fU, 0xce141879U, 0x37c773bfU,
        0xcdf753eaU, 0xaafd5f5bU, 0x6f3ddf14U, 0xdb447886U,
        0xf3afca81U, 0xc468b93eU, 0x3424382cU, 0x40a3c25fU,
        0xc31d1672U, 0x25e2bc0cU, 0x493c288bU, 0x950dff41U,
        0x01a83971U, 0xb30c08deU, 0xe4b4d89cU, 0xc1566490U,
        0x84cb7b61U, 0xb632d570U, 0x5c6c4874U, 0x57b8d042U,

    };

    int T3Temp[256] = {

        0xf45150a7U, 0x417e5365U, 0x171ac3a4U, 0x273a965eU,
        0xab3bcb6bU, 0x9d1ff145U, 0xfaacab58U, 0xe34b9303U,
        0x302055faU, 0x76adf66dU, 0xcc889176U, 0x02f5254cU,
        0xe54ffcd7U, 0x2ac5d7cbU, 0x35268044U, 0x62b58fa3U,
        0xb1de495aU, 0xba25671bU, 0xea45980eU, 0xfe5de1c0U,
        0x2fc30275U, 0x4c8112f0U, 0x468da397U, 0xd36bc6f9U,
        0x8f03e75fU, 0x9215959cU, 0x6dbfeb7aU, 0x5295da59U,
        0xbed42d83U, 0x7458d321U, 0xe0492969U, 0xc98e44c8U,
        0xc2756a89U, 0x8ef47879U, 0x58996b3eU, 0xb927dd71U,
        0xe1beb64fU, 0x88f017adU, 0x20c966acU, 0xce7db43aU,
        0xdf63184aU, 0x1ae58231U, 0x51976033U, 0x5362457fU,
        0x64b1e077U, 0x6bbb84aeU, 0x81fe1ca0U, 0x08f9942bU,
        0x48705868U, 0x458f19fdU, 0xde94876cU, 0x7b52b7f8U,
        0x73ab23d3U, 0x4b72e202U, 0x1fe3578fU, 0x55662aabU,
        0xebb20728U, 0xb52f03c2U, 0xc5869a7bU, 0x37d3a508U,
        0x2830f287U, 0xbf23b2a5U, 0x0302ba6aU, 0x16ed5c82U,
        0xcf8a2b1cU, 0x79a792b4U, 0x07f3f0f2U, 0x694ea1e2U,
        0xda65cdf4U, 0x0506d5beU, 0x34d11f62U, 0xa6c48afeU,
        0x2e349d53U, 0xf3a2a055U, 0x8a0532e1U, 0xf6a475ebU,
        0x830b39ecU, 0x6040aaefU, 0x715e069fU, 0x6ebd5110U,
        0x213ef98aU, 0xdd963d06U, 0x3eddae05U, 0xe64d46bdU,
        0x5491b58dU, 0xc471055dU, 0x06046fd4U, 0x5060ff15U,
        0x981924fbU, 0xbdd697e9U, 0x4089cc43U, 0xd967779eU,
        0xe8b0bd42U, 0x8907888bU, 0x19e7385bU, 0xc879dbeeU,
        0x7ca1470aU, 0x427ce90fU, 0x84f8c91eU, 0x00000000U,
        0x80098386U, 0x2b3248edU, 0x111eac70U, 0x5a6c4e72U,
        0x0efdfbffU, 0x850f5638U, 0xae3d1ed5U, 0x2d362739U,
        0x0f0a64d9U, 0x5c6821a6U, 0x5b9bd154U, 0x36243a2eU,
        0x0a0cb167U, 0x57930fe7U, 0xeeb4d296U, 0x9b1b9e91U,
        0xc0804fc5U, 0xdc61a220U, 0x775a694bU, 0x121c161aU,
        0x93e20abaU, 0xa0c0e52aU, 0x223c43e0U, 0x1b121d17U,
        0x090e0b0dU, 0x8bf2adc7U, 0xb62db9a8U, 0x1e14c8a9U,
        0xf1578519U, 0x75af4c07U, 0x99eebbddU, 0x7fa3fd60U,
        0x01f79f26U, 0x725cbcf5U, 0x6644c53bU, 0xfb5b347eU,
        0x438b7629U, 0x23cbdcc6U, 0xedb668fcU, 0xe4b863f1U,
        0x31d7cadcU, 0x63421085U, 0x97134022U, 0xc6842011U,
        0x4a857d24U, 0xbbd2f83dU, 0xf9ae1132U, 0x29c76da1U,
        0x9e1d4b2fU, 0xb2dcf330U, 0x860dec52U, 0xc177d0e3U,
        0xb32b6c16U, 0x70a999b9U, 0x9411fa48U, 0xe9472264U,
        0xfca8c48cU, 0xf0a01a3fU, 0x7d56d82cU, 0x3322ef90U,
        0x4987c74eU, 0x38d9c1d1U, 0xca8cfea2U, 0xd498360bU,
        0xf5a6cf81U, 0x7aa528deU, 0xb7da268eU, 0xad3fa4bfU,
        0x3a2ce49dU, 0x78500d92U, 0x5f6a9bccU, 0x7e546246U,
        0x8df6c213U, 0xd890e8b8U, 0x392e5ef7U, 0xc382f5afU,
        0x5d9fbe80U, 0xd0697c93U, 0xd56fa92dU, 0x25cfb312U,
        0xacc83b99U, 0x1810a77dU, 0x9ce86e63U, 0x3bdb7bbbU,
        0x26cd0978U, 0x596ef418U, 0x9aec01b7U, 0x4f83a89aU,
        0x95e6656eU, 0xffaa7ee6U, 0xbc2108cfU, 0x15efe6e8U,
        0xe7bad99bU, 0x6f4ace36U, 0x9fead409U, 0xb029d67cU,
        0xa431afb2U, 0x3f2a3123U, 0xa5c63094U, 0xa235c066U,
        0x4e7437bcU, 0x82fca6caU, 0x90e0b0d0U, 0xa73315d8U,
        0x04f14a98U, 0xec41f7daU, 0xcd7f0e50U, 0x91172ff6U,
        0x4d768dd6U, 0xef434db0U, 0xaacc544dU, 0x96e4df04U,
        0xd19ee3b5U, 0x6a4c1b88U, 0x2cc1b81fU, 0x65467f51U,
        0x5e9d04eaU, 0x8c015d35U, 0x87fa7374U, 0x0bfb2e41U,
        0x67b35a1dU, 0xdb9252d2U, 0x10e93356U, 0xd66d1347U,
        0xd79a8c61U, 0xa1377a0cU, 0xf8598e14U, 0x13eb893cU,
        0xa9ceee27U, 0x61b735c9U, 0x1ce1ede5U, 0x477a3cb1U,
        0xd29c59dfU, 0xf2553f73U, 0x141879ceU, 0xc773bf37U,
        0xf753eacdU, 0xfd5f5baaU, 0x3ddf146fU, 0x447886dbU,
        0xafca81f3U, 0x68b93ec4U, 0x24382c34U, 0xa3c25f40U,
        0x1d1672c3U, 0xe2bc0c25U, 0x3c288b49U, 0x0dff4195U,
        0xa8397101U, 0x0c08deb3U, 0xb4d89ce4U, 0x566490c1U,
        0xcb7b6184U, 0x32d570b6U, 0x6c48745cU, 0xb8d04257U,

       };
    int T4Temp[256] = {

        0x5150a7f4U, 0x7e536541U, 0x1ac3a417U, 0x3a965e27U,
        0x3bcb6babU, 0x1ff1459dU, 0xacab58faU, 0x4b9303e3U,
        0x2055fa30U, 0xadf66d76U, 0x889176ccU, 0xf5254c02U,
        0x4ffcd7e5U, 0xc5d7cb2aU, 0x26804435U, 0xb58fa362U,
        0xde495ab1U, 0x25671bbaU, 0x45980eeaU, 0x5de1c0feU,
        0xc302752fU, 0x8112f04cU, 0x8da39746U, 0x6bc6f9d3U,
        0x03e75f8fU, 0x15959c92U, 0xbfeb7a6dU, 0x95da5952U,
        0xd42d83beU, 0x58d32174U, 0x492969e0U, 0x8e44c8c9U,
        0x756a89c2U, 0xf478798eU, 0x996b3e58U, 0x27dd71b9U,
        0xbeb64fe1U, 0xf017ad88U, 0xc966ac20U, 0x7db43aceU,
        0x63184adfU, 0xe582311aU, 0x97603351U, 0x62457f53U,
        0xb1e07764U, 0xbb84ae6bU, 0xfe1ca081U, 0xf9942b08U,
        0x70586848U, 0x8f19fd45U, 0x94876cdeU, 0x52b7f87bU,
        0xab23d373U, 0x72e2024bU, 0xe3578f1fU, 0x662aab55U,
        0xb20728ebU, 0x2f03c2b5U, 0x869a7bc5U, 0xd3a50837U,
        0x30f28728U, 0x23b2a5bfU, 0x02ba6a03U, 0xed5c8216U,
        0x8a2b1ccfU, 0xa792b479U, 0xf3f0f207U, 0x4ea1e269U,
        0x65cdf4daU, 0x06d5be05U, 0xd11f6234U, 0xc48afea6U,
        0x349d532eU, 0xa2a055f3U, 0x0532e18aU, 0xa475ebf6U,
        0x0b39ec83U, 0x40aaef60U, 0x5e069f71U, 0xbd51106eU,
        0x3ef98a21U, 0x963d06ddU, 0xddae053eU, 0x4d46bde6U,
        0x91b58d54U, 0x71055dc4U, 0x046fd406U, 0x60ff1550U,
        0x1924fb98U, 0xd697e9bdU, 0x89cc4340U, 0x67779ed9U,
        0xb0bd42e8U, 0x07888b89U, 0xe7385b19U, 0x79dbeec8U,
        0xa1470a7cU, 0x7ce90f42U, 0xf8c91e84U, 0x00000000U,
        0x09838680U, 0x3248ed2bU, 0x1eac7011U, 0x6c4e725aU,
        0xfdfbff0eU, 0x0f563885U, 0x3d1ed5aeU, 0x3627392dU,
        0x0a64d90fU, 0x6821a65cU, 0x9bd1545bU, 0x243a2e36U,
        0x0cb1670aU, 0x930fe757U, 0xb4d296eeU, 0x1b9e919bU,
        0x804fc5c0U, 0x61a220dcU, 0x5a694b77U, 0x1c161a12U,
        0xe20aba93U, 0xc0e52aa0U, 0x3c43e022U, 0x121d171bU,
        0x0e0b0d09U, 0xf2adc78bU, 0x2db9a8b6U, 0x14c8a91eU,
        0x578519f1U, 0xaf4c0775U, 0xeebbdd99U, 0xa3fd607fU,
        0xf79f2601U, 0x5cbcf572U, 0x44c53b66U, 0x5b347efbU,
        0x8b762943U, 0xcbdcc623U, 0xb668fcedU, 0xb863f1e4U,
        0xd7cadc31U, 0x42108563U, 0x13402297U, 0x842011c6U,
        0x857d244aU, 0xd2f83dbbU, 0xae1132f9U, 0xc76da129U,
        0x1d4b2f9eU, 0xdcf330b2U, 0x0dec5286U, 0x77d0e3c1U,
        0x2b6c16b3U, 0xa999b970U, 0x11fa4894U, 0x472264e9U,
        0xa8c48cfcU, 0xa01a3ff0U, 0x56d82c7dU, 0x22ef9033U,
        0x87c74e49U, 0xd9c1d138U, 0x8cfea2caU, 0x98360bd4U,
        0xa6cf81f5U, 0xa528de7aU, 0xda268eb7U, 0x3fa4bfadU,
        0x2ce49d3aU, 0x500d9278U, 0x6a9bcc5fU, 0x5462467eU,
        0xf6c2138dU, 0x90e8b8d8U, 0x2e5ef739U, 0x82f5afc3U,
        0x9fbe805dU, 0x697c93d0U, 0x6fa92dd5U, 0xcfb31225U,
        0xc83b99acU, 0x10a77d18U, 0xe86e639cU, 0xdb7bbb3bU,
        0xcd097826U, 0x6ef41859U, 0xec01b79aU, 0x83a89a4fU,
        0xe6656e95U, 0xaa7ee6ffU, 0x2108cfbcU, 0xefe6e815U,
        0xbad99be7U, 0x4ace366fU, 0xead4099fU, 0x29d67cb0U,
        0x31afb2a4U, 0x2a31233fU, 0xc63094a5U, 0x35c066a2U,
        0x7437bc4eU, 0xfca6ca82U, 0xe0b0d090U, 0x3315d8a7U,
        0xf14a9804U, 0x41f7daecU, 0x7f0e50cdU, 0x172ff691U,
        0x768dd64dU, 0x434db0efU, 0xcc544daaU, 0xe4df0496U,
        0x9ee3b5d1U, 0x4c1b886aU, 0xc1b81f2cU, 0x467f5165U,
        0x9d04ea5eU, 0x015d358cU, 0xfa737487U, 0xfb2e410bU,
        0xb35a1d67U, 0x9252d2dbU, 0xe9335610U, 0x6d1347d6U,
        0x9a8c61d7U, 0x377a0ca1U, 0x598e14f8U, 0xeb893c13U,
        0xceee27a9U, 0xb735c961U, 0xe1ede51cU, 0x7a3cb147U,
        0x9c59dfd2U, 0x553f73f2U, 0x1879ce14U, 0x73bf37c7U,
        0x53eacdf7U, 0x5f5baafdU, 0xdf146f3dU, 0x7886db44U,
        0xca81f3afU, 0xb93ec468U, 0x382c3424U, 0xc25f40a3U,
        0x1672c31dU, 0xbc0c25e2U, 0x288b493cU, 0xff41950dU,
        0x397101a8U, 0x08deb30cU, 0xd89ce4b4U, 0x6490c156U,
        0x7b6184cbU, 0xd570b632U, 0x48745c6cU, 0xd04257b8U,

    };
    for(int i=0; i<256;i++){
        matrizCajaS[i] = matrizCajaSTemp[i];
        T1[i] = T1Temp[i];
        T2[i] = T2Temp[i];
        T3[i] = T3Temp[i];
        T4[i] = T4Temp[i];
    }
    return 1;
}
__device__ void AES_128(aesBlock *m, unsigned int *keys, int index, unsigned char *matrizCajaS, int *T1, int *T2, int *T3, int *T4){

         int shifttab[16]= {
            0, 5, 10, 15,
            4,  9,  14,  3,
            8,  13,  2,  7,
            12,  1,  6,  11,
        };

        // imprimiArregloCuda(16,(unsigned char *)&m[index].block );


        addRoundKey( (unsigned int*)&m[index].block, keys,0);


        for (int j = 1; j < 10; j++){
            // subBytes(block, matrizCajaS);
            shiftRows((unsigned int*)&m[index].block, shifttab);

            //mixColumns(block);
            subBytesMixColumns((unsigned int*)&m[index].block,  T1,  T2,  T3,  T4);

            //añadimos llave de ronda
            addRoundKey( (unsigned int*)&m[index].block, keys,j); //
        }
        // imprimiArregloCuda(16,(unsigned char *)&m[index].block );
        // imprimiArregloCuda(16,(unsigned char *)&m[index].block );

        subBytes((unsigned int*)&m[index].block, matrizCajaS);
        // imprimiArregloCuda(16,(unsigned char *)&m[index].block );

        shiftRows((unsigned int*)&m[index].block, shifttab);
        // imprimiArregloCuda(16,(unsigned char *)&m[index].block );

        addRoundKey( (unsigned int*)&m[index].block, keys,10);
        // imprimiArregloCuda(16,(unsigned char *)&m[index].block );

        // imprimiArregloCuda(16,(unsigned char *)&keys[40] );


}

__device__ void OCBAESDelta2Rounds(unsigned int block[4], unsigned int *keys, unsigned char *matrizCajaS, int *T1, int *T2, int *T3, int *T4){
         int shifttab[16]= {
            0, 5, 10, 15,
            4,  9,  14,  3,
            8,  13,  2,  7,
            12,  1,  6,  11,
        };

        for (int j = 1; j < 3; j++){
            shiftRows(block, shifttab);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);
            addRoundKey( block, keys,j); //
        }
}

__device__ void AES_128Decrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys, int index ){
    // __shared__ unsigned char matrizCajaS[256];
    // __shared__ int T1[256];
    // __shared__ int T2[256];
    // __shared__ int T3[256];
    // __shared__ int T4[256];

    unsigned char matrizCajaS[256];
    int T1[256];
    int T2[256];
    int T3[256];
    int T4[256];
        // if(threadIdx.x == 0 || index==0 ){
            AES_init_decrypt(matrizCajaS, T1, T2, T3, T4);
        // }

            int shifttab[16]= {
                0, 13,  10,  7,
                4,  1,   14,  11,
                8,  5,   2,  15,
                12,  9,   6,  3,
            };

        // __syncthreads();
        unsigned int block[4];

        for (int i = 0 ; i< 4 ; i++){
            block[i]= m[index].block[i];
        }

        addRoundKey( block, keys,0);



        for (int j = 1; j < 10; j++){
            // subBytes(block, matrizCajaS);
            shiftRows(block, shifttab);

            //mixColumns(block);
            subBytesMixColumns(block,  T1,  T2,  T3,  T4);

            //añadimos llave de ronda
            addRoundKey( block, keys,j); //

        }

        subBytes(block, matrizCajaS);

        shiftRows(block, shifttab);

        addRoundKey( block, keys,10);
        for (int i = 0 ; i< 4 ; i++){
            m[index].block[i]= block[i];
        }

}

__global__ void Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys){
    
    int index = blockDim.x*blockIdx.x + threadIdx.x;

    //thread_block g = this_thread_block();
    __shared__ unsigned char matrizCajaS[256];
    __shared__ int T1[256];
    __shared__ int T2[256];
    __shared__ int T3[256];
    __shared__ int T4[256];

        if(threadIdx.x <256  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez

            T1[threadIdx.x] = T1Temp[threadIdx.x];
            T2[threadIdx.x] = T2Temp[threadIdx.x];
            T3[threadIdx.x] = T3Temp[threadIdx.x];
            T4[threadIdx.x] = T4Temp[threadIdx.x];
            matrizCajaS[threadIdx.x] = matrizCajaSTemp[threadIdx.x];

        }

    __syncthreads();
    int totalThreads = NumberBlocks * NumnerThreads;
    int totalCyclesFor;
    if(mlen%totalThreads == 0)
        totalCyclesFor = mlen/totalThreads;
    else
        totalCyclesFor = mlen/totalThreads + 1;

    for (int k = 0; k < totalCyclesFor; k ++){
        if( index<mlen){
            AES_128(m, keys, index, matrizCajaS,T1, T2, T3, T4 );
        }
        index = index + totalThreads;
    }
}



void AES128Encrypt(aesBlock *m, unsigned long long mlen, unsigned int *keys){
    aesBlock *mCuda;
    unsigned int *keysCuda;

    int sizeMessage = (mlen)*sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);

    dim3 nb( NumberBlocks ) ;
    dim3 nt(NumnerThreads);

    Encrypt<<<nb, nt>>>(mCuda, mlen,keysCuda);

    hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);

    hipFree(mCuda); hipFree(keysCuda);
}


__global__ void OCB128EncryptRandomAccess(aesBlock *m, aesBlock *result, aesBlock *delta, aesBlock *S,  unsigned long long mlen,unsigned long long mlenReal, unsigned long long deltalen, unsigned int *keys){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    cg::grid_group g = cg::this_grid(); //para sincronizar el grid es decir todos los hilos de la tarjeta

    //thread_block g = this_thread_block();
    __shared__ unsigned char matrizCajaS[256];
    __shared__ int T1[256];
    __shared__ int T2[256];
    __shared__ int T3[256];
    __shared__ int T4[256];

        if(threadIdx.x <256  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez

            T1[threadIdx.x] = T1Temp[threadIdx.x];
            T2[threadIdx.x] = T2Temp[threadIdx.x];
            T3[threadIdx.x] = T3Temp[threadIdx.x];
            T4[threadIdx.x] = T4Temp[threadIdx.x];
            matrizCajaS[threadIdx.x] = matrizCajaSTemp[threadIdx.x];

        }
    
   
      
     int shifttabLE[16]= {
            3, 2, 1, 0,
            7,  6,  5,  4,
            11,  10,  9,  8,
            15,  14,  13,  12,
        };
    int shifttabBE[16]= {
        0, 1, 2, 3,
        4,  5,  6,  7,
        8,  9,  10,  11,
        12,  13,  14,  15,
    };


    __syncthreads();
    int totalThreads = NumberBlocks * NumnerThreads;
    int totalCyclesFor;
    if(mlen%totalThreads == 0)
        totalCyclesFor = mlen/totalThreads;
    else
        totalCyclesFor = mlen/totalThreads + 1;

    unsigned int deltaBlock[4];
     
    
    for (int k = 0; k < totalCyclesFor; k ++){
        if(index == mlen-1){
            break;
        }
        for (int i = 0 ; i< 4 ; i++){
            deltaBlock[i]= delta[0].block[i];
        }
        shiftRows(deltaBlock, shifttabLE);        

        if( index<mlen-1){
            atomicXor(&m[mlen-1].block[0], m[index].block[0]);
            atomicXor(&m[mlen-1].block[1], m[index].block[1]);
            atomicXor(&m[mlen-1].block[2], m[index].block[2]);
            atomicXor(&m[mlen-1].block[3], m[index].block[3]);
            // XOR_128(m[mlen-1].block,m[index].block);
            if(index == (mlen - 2)){
                if(mlenReal%16==0){
                    // imprimiArregloCuda(16,(unsigned char *)&m[index].block );
                    // imprimiArregloCuda(16,(unsigned char *)&delta[index].block );
                    // imprimiArregloCuda(16,(unsigned char *)&delta[index-1].block );
                    // imprimiArregloCuda(16,(unsigned char *)&m[index].block );
                    //carga del valor de delta

                    for (int i = 0 ; i< 4 ; i++){
                        deltaBlock[i]= deltaBlock[i]+index;
                    }
                    shiftRows(deltaBlock, shifttabLE);        

                    //Calculo de dos rondas de aes
                    OCBAESDelta2Rounds(deltaBlock, keys, matrizCajaS, T1,T2,T3,T4);
                    XOR_128(m[index].block,deltaBlock);
                    AES_128(m, keys,index, matrizCajaS, T1,T2,T3,T4);
                    XOR2_128(result[index].block, m[index].block,deltaBlock );

                }else{
                    aesBlock *aestemp;
                    aestemp = new aesBlock [1];
                    for (int i = 0 ; i< 4 ; i++){
                        aestemp[0].block[i] = deltaBlock[i]+index+1;
                    }
                    shiftRows(aestemp[0].block, shifttabLE);        

                    OCBAESDelta2Rounds(aestemp[0].block, keys, matrizCajaS, T1,T2,T3,T4);
                    AES_128(aestemp, keys,0,matrizCajaS, T1,T2,T3,T4);
                    XOR2_128(result[index].block, m[index].block,deltaBlock );
                    
                }
            }
            else{
                //carga del valor de delta
                for (int i = 0 ; i< 4 ; i++){
                    deltaBlock[i]= deltaBlock[i]+index;
                }
                shiftRows(deltaBlock, shifttabLE);        

                //Calculo de dos rondas de aes
                OCBAESDelta2Rounds(deltaBlock, keys, matrizCajaS, T1,T2,T3,T4);
                XOR_128(m[index].block,deltaBlock);
                AES_128(m, keys,index, matrizCajaS, T1,T2,T3,T4);
                XOR2_128(result[index].block, m[index].block,deltaBlock );
            }
        }
        index = index + totalThreads;
    }
    g.sync(); //cambiar por el g.sync
    if( (index)==mlen-1){
        for (int i = 0 ; i< 4 ; i++){
            deltaBlock[i]= delta[0].block[i];
        }
        
        imprimiArregloCudaInt(4, deltaBlock);
        // imprimiArregloCudaInt(4, delta[0].block);
        shiftRows(deltaBlock, shifttabLE);        
        imprimiArregloCudaInt(4, deltaBlock);

        for (int i = 0 ; i< 4 ; i++){
            deltaBlock[i]= deltaBlock[i]+index+3;
        }
        shiftRows(deltaBlock, shifttabLE);        
         
        imprimiArregloCudaInt(4, deltaBlock);
        // imprimiArregloCuda(16,(unsigned char *)&deltaBlock);
        // imprimiArregloCudaInt(4, deltaBlock);
        OCBAESDelta2Rounds(deltaBlock, keys, matrizCajaS, T1,T2,T3,T4);

        XOR_128(m[mlen-1].block,deltaBlock);
        AES_128(m, keys,mlen-1, matrizCajaS, T1,T2,T3,T4);
        XOR2_128(result[mlen-1].block, m[mlen-1].block,S[0].block );
    }


}



__global__ void OCB128DecryptRandomAcces(aesBlock *m, aesBlock *result, aesBlock *delta, aesBlock *S,  unsigned long long mlen,unsigned long long mlenReal, unsigned long long deltalen, unsigned int *keys, unsigned int *encrypt_keys ){
    int index = blockDim.x*blockIdx.x + threadIdx.x;

    __shared__ unsigned char matrizCajaS[256];
    __shared__ int T1[256];
    __shared__ int T2[256];
    __shared__ int T3[256];
    __shared__ int T4[256];

    if(threadIdx.x == 0  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez
        for(int i=0; i<256;i++){
            matrizCajaS[i] = matrizCajaSTemp[i];
        }
    }

    if(threadIdx.x == 1  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez
        for(int i=0; i<256;i++){
            T1[i] = T1Temp[i];
        }
    }
    if(threadIdx.x == 2  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez
        for(int i=0; i<256;i++){
            T2[i] = T2Temp[i];
        }
    }
    if(threadIdx.x == 3  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez
        for(int i=0; i<256;i++){
            T3[i] = T3Temp[i];
        }
    }
    if(threadIdx.x == 4  ){ //le damos chance de que cada instancia logre inicializar cada valor minimo una vez
        for(int i=0; i<256;i++){
            T4[i] = T4Temp[i];
        }
    }
    __syncthreads();

    if( index<mlen){
        __syncthreads();
        if(index == (mlen - 1)){
            if(mlenReal%16==0){
                XOR_128(m[index].block,delta[index].block);
                AES_128Decrypt(m, mlen, keys,index);
                XOR_128(m[index].block,delta[index].block);

            }else{
                aesBlock *aesTemp = new aesBlock [1];

                aesTemp[0].block[0] = delta[index].block[0];
                aesTemp[0].block[1] = delta[index].block[1];
                aesTemp[0].block[2] = delta[index].block[2];
                aesTemp[0].block[3] = delta[index].block[3];
                AES_128( aesTemp, encrypt_keys,0,matrizCajaS, T1,T2,T3,T4);

                XOR_128(m[index].block,aesTemp[0].block);
            }

        }
        else{


            XOR_128(m[index].block,delta[index].block);

            AES_128Decrypt(m, mlen, keys,index);

            XOR_128(m[index].block,delta[index].block);

        }
    }
}

void OCBRandomAccess(aesBlock *m,aesBlock *delta, aesBlock *S, unsigned long long message_len, const unsigned long long mlenReal, unsigned long long deltalen, unsigned int *keys,unsigned int *decrypt_keys, int encrypt){
    aesBlock *mCuda;
    aesBlock *resultCuda;
    aesBlock *SCuda;
    aesBlock *deltaCuda;
    unsigned int *keysCuda;
    unsigned int *decrypt_keysCuda;

    int sizeMessage = (message_len)*sizeof(class aesBlock);
    int sizeDelta = (deltalen)*sizeof(class aesBlock);
    int sizeS = sizeof(class aesBlock);
    int sizeKeys = 11*4*sizeof(unsigned int);

    hipMalloc(&mCuda, sizeMessage);
    hipMalloc(&resultCuda, sizeMessage);
    hipMalloc(&keysCuda, sizeKeys);
    hipMalloc(&decrypt_keysCuda, sizeKeys);
    hipMalloc(&deltaCuda, sizeDelta);
    hipMalloc(&SCuda, sizeS);

    hipMemcpy(mCuda,m,sizeMessage,hipMemcpyDefault);
    hipMemcpy(deltaCuda,delta,sizeDelta,hipMemcpyDefault);
    hipMemcpy(keysCuda,keys,sizeKeys,hipMemcpyDefault);
    hipMemcpy(decrypt_keysCuda,decrypt_keys,sizeKeys,hipMemcpyDefault);
    hipMemcpy(SCuda,S,sizeS,hipMemcpyDefault);
    // printf("%f \n", ceil( (mlen+1)/4.0 ));

    int device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    void *kernelArgs[] = { (void *)&mCuda,(void *)&resultCuda,(void *)&deltaCuda,(void *)&SCuda,(void *)&message_len, (void *)&mlenReal, (void *)&deltalen, (void *)&keysCuda};
    // aesBlock *m, aesBlock *result, aesBlock *delta, aesBlock *S,  unsigned long long mlen,unsigned long long mlenReal, unsigned long long deltalen, unsigned int *keys
    dim3 nb(NumberBlocks,1,1) ;
    dim3 nt(NumnerThreads,1,1);

    size_t smem = sizeof(int) * (1 << 11);
    //if(encrypt)
        gpuchk(hipLaunchCooperativeKernel((void*)OCB128EncryptRandomAccess, nb, nt, kernelArgs, smem, NULL));
        //OCB128EncryptRandomAccess<<<nb, nt>>>(mCuda,resultCuda,deltaCuda,SCuda, message_len, mlenReal, deltalen,keysCuda);
    // else
        // OCB128DecryptRandomAcces<<<nb, nt>>>(mCuda, resultCuda,deltaCuda, SCuda,message_len, mlenReal, deltalen,decrypt_keysCuda, keysCuda);

    // hipMemcpy(m, mCuda, sizeMessage, hipMemcpyDefault);
    hipMemcpy(m, resultCuda, sizeMessage, hipMemcpyDefault);

    hipFree(SCuda);
    hipFree(mCuda);
    hipFree(keysCuda);
    hipFree(deltaCuda);
    hipFree(resultCuda);
    hipFree(decrypt_keysCuda);
}



void getDelta(const unsigned char *nsec, aesBlock* delta,unsigned int *keys,unsigned long long deltalen ){
    // unsigned int nonce[4] = {0,};
     aesBlock *nonce = new aesBlock[1];
    copyMessageToAESBlock(nonce, 1, (unsigned int *)&nsec[0]);
    
    // unsignedCharArrayTounsignedIntArray(nsec,nonce,16,16);

    for(int i = 0; i<deltalen; i++){

        for (int j = 0; j<4;j++){
                delta[i].block[j]= nonce[i].block[j]+i;
        }
    }
    AES128Encrypt(delta, deltalen, keys);

}

void copyMessageToAESBlock(aesBlock* encrypt, int numBlocks, unsigned int * m2 ){
    for(int i = 0; i<numBlocks; i++){
        for (int j = 0; j<4;j++){
            encrypt[i].block[j]=  m2[(i*4)+j];
        }
    }
}

void copyAESBlockToMessage(aesBlock* encrypt, int numBlocks, unsigned int * m2){
    for(int i = 0; i<numBlocks; i++){
        for (int j = 0; j<4;j++){
            m2[(i*4)+j]=encrypt[i].block[j];
        }
    }
}

void unsignedCharArrayTounsignedIntArray(const unsigned char *in,unsigned int *out, unsigned long long len, unsigned long long mlen2 )
{

    unsigned char h[mlen2]={0};
    unsigned char temp[mlen2]={0};

    memcpy(h, in, len);
    memcpy(temp, in, mlen2);

    int shifttab[16]= {
        3, 2, 1, 0,
        7, 6, 5, 4,
        11, 10, 9, 8,
        15, 14, 13, 12
        };
    if(len%16!=0){
        h[len]=0x01;
    }

    for(int i = 0; i < mlen2; i++){
        int index = shifttab[i%16]+(floor(i/16)*16 );
        temp[i] = h[index];
    }

    unsigned int * temp2;
    temp2 = (unsigned int *) temp;

    for(int i = 0; i < mlen2/4; i++){
        out[i]=temp2[i];
    }


}


static void process_ad(ae_ctx *ctx, aesBlock *S, aesBlock *delta_ad ,unsigned char *ad, int ad_len, int final){

    int ad_len_temp = ad_len/16;
    int delta_len = ad_len/16;
    if(ad_len%16!=0){
        delta_len++;
        ad_len_temp++;
    }

    aesBlock *message_ad = new aesBlock[delta_len];
    copyMessageToAESBlock(message_ad, ad_len_temp, (unsigned int *)&ad[0]);
    // OCBRandomAccessAsociatedData(message_ad, delta_ad, ad_len_temp, ad_len, &ctx->encrypt_key.keys[0][0]);
    // imprimiArreglo(16,(unsigned char *)& S[0].block );
}


int crypto_aead_encrypt(
	unsigned char *c, unsigned long long *clen,
	unsigned char *m, unsigned long long mlen,
	unsigned char *ad, unsigned long long ad_len,
	unsigned char *t,
	const unsigned char *nsec,
	const unsigned char *npub,
	const unsigned char *k)
{

    int message_len = mlen/16+1;
    unsigned int delta_len = ( (mlen/16)+1 )/(2147483648);
    if(mlen%16!=0){
        message_len++;
    }
    if (delta_len==0)
    {
        delta_len++;
    }
    

    int ad_len_temp = ad_len/16;
    if(ad_len%16!=0){
        ad_len_temp++;
    }

    ae_ctx* ctx = ae_allocate(NULL);

    aesBlock *delta = new aesBlock[delta_len];
    aesBlock *message = new aesBlock[message_len];
    aesBlock *S = new aesBlock[1];
    aesBlock *delta_ad = new aesBlock[ad_len_temp];
    ae_init(ctx, k, 16, 12, 16);
    if (nsec) {
        // getDelta(ctx,ctx->ad_offset, delta_ad, ad_len, 1);
        getDelta(nsec, delta, &ctx->encrypt_key.keys[0][0], delta_len );
    }

    if (ad_len > 0){
        process_ad(ctx, S, delta_ad,ad,  ad_len, 1);
    }
    // imprimiArreglo(16,(unsigned char *)&delta[0]);
    // exit(1);
    copyMessageToAESBlock(message, message_len-1, (unsigned int *)&m[0]);
    // checksum (message, message_len-1, mlen, message[message_len-1].block );
    OCBRandomAccess(message, delta, S, message_len, mlen, delta_len, &ctx->encrypt_key.keys[0][0], &ctx->decrypt_key.keys[0][0],1);
    copyAESBlockToMessage(message, message_len, (unsigned int *)&c[0]);
    copyAESBlockToMessage(&message[message_len-1],1, (unsigned int *)&t[0]);
    // imprimiArreglo(16,(unsigned char *)&ctx->offset);
    return 1;
}





const unsigned long long mlen=1048576;
unsigned char m[mlen+16] ={0};
unsigned char c[mlen+16]={0};

int main(int argc, char **argv) {
    const unsigned char k[16] ={
        0x2b, 0x7e, 0x15, 0x16,
        0x28, 0xae, 0xd2, 0xa6,
        0xab, 0xf7, 0x15, 0x88,
        0x09, 0xcf, 0x4f, 0x3c,

    };



    unsigned char Tag[16] ={0,};
    unsigned long long adlen = 0;

    unsigned char ad[adlen];

    for(int j=0;j<mlen; j++){
        m[j]=j;
    }
    for(int j=0;j<adlen; j++){
        ad[j]=j;
    }
    unsigned long long *clen;


     unsigned char nsec[16] = {
        0x32, 0x43, 0xf6, 0xa8,
        0x88, 0x5a, 0x30, 0x8d,
        0x31, 0x31, 0x98, 0xa2,
        0xe0, 0x37, 0x07, 0x35,
    };
    for(int j=0;j<16; j++){
        nsec[j]=j;
    }
    const unsigned char *npub;

    crypto_aead_encrypt(c, clen, m, mlen, ad, adlen,Tag, nsec, npub, k);
    cout<<"\n---------------------------Encrypt------------------------------------         \n";

    printf("\nlen: %lli\n",mlen);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Key          ";
    imprimiArreglo(16,k);
    printf("\n---------------------------");
    cout<<endl;


    cout<<"Nonce        ";
    imprimiArreglo(16,nsec);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Plaintext    ";
    print_hex_string(m,mlen);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Ciphertext   ";
    print_hex_string(c,mlen);
    printf("\n---------------------------");
    cout<<endl;

    cout<<"Tag          ";
    imprimiArreglo(16,(unsigned char *)&Tag[0]);
    cout<<endl;


    int dev = 0;
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
    cout <<"supportsCoopLaunch "<<supportsCoopLaunch<< endl;
   if (__cplusplus == 201703L)
        std::cout << "C++17" << endl;
    else if (__cplusplus == 201402L)
        std::cout << "C++14" << endl;
    else if (__cplusplus == 201103L)
        std::cout << "C++11" << endl;
    else if (__cplusplus == 199711L)
        std::cout << "C++98" << endl;
    else
        std::cout << "pre-standard C++" << endl;
    // for(int j=0;j<adlen; j++){
    //     ad[j]=j;
    // }
    // crypto_aead_decrypt(m, clen, c, mlen, ad, adlen, nsec, npub, k);

    return 0;
}
